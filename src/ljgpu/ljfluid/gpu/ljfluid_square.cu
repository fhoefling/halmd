#include "hip/hip_runtime.h"
/* Lennard-Jones fluid kernel
 *
 * Copyright © 2008-2009  Peter Colberg
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <ljgpu/ljfluid/gpu/base.cuh>
#include <ljgpu/ljfluid/gpu/ljfluid_square.hpp>

namespace ljgpu { namespace gpu
{

/**
 * MD simulation step
 */
template <typename T, typename TT, typename U>
__global__ void mdstep(U* g_r, U* g_v, U* g_f, float* g_en, float* g_virial)
{
    extern __shared__ T s_r[];

    // load particle associated with this thread
    T r = unpack(g_r[GTID]);
    T v = unpack(g_v[GTID]);

    // potential energy contribution
    float en = 0;
    // virial equation sum contribution
    float virial = 0;
    // force sum
    TT f = 0;

    // iterate over all blocks
    for (unsigned int k = 0; k < gridDim.x; k++) {
	// load positions of particles within block
	s_r[TID] = unpack(g_r[k * blockDim.x + TID]);
	__syncthreads();

	// iterate over all particles within block
	for (unsigned int j = 0; j < blockDim.x; j++) {
	    // skip placeholder particles
	    if (k * blockDim.x + j >= npart)
		continue;
	    // skip identical particle
	    if (blockIdx.x == k && TID == j)
		continue;

	    // compute Lennard-Jones force with particle
	    compute_force(r, s_r[j], f, en, virial);
	}
	__syncthreads();
    }

    // second leapfrog step of integration of equations of motion
    leapfrog_full_step(v, f.f0);

    // store particle associated with this thread
    g_v[GTID] = pack(v);
    g_f[GTID] = pack(f.f0);
    g_en[GTID] = en;
    g_virial[GTID] = virial;
}

/**
 * device constant wrappers
 */
cuda::symbol<uint> ljfluid_base<ljfluid_impl_gpu_square>::npart(gpu::npart);
cuda::symbol<float> ljfluid_base<ljfluid_impl_gpu_square>::box(gpu::box);
cuda::symbol<float> ljfluid_base<ljfluid_impl_gpu_square>::timestep(gpu::timestep);
cuda::symbol<float> ljfluid_base<ljfluid_impl_gpu_square>::r_cut(gpu::r_cut);
cuda::symbol<float> ljfluid_base<ljfluid_impl_gpu_square>::rr_cut(gpu::rr_cut);
cuda::symbol<float> ljfluid_base<ljfluid_impl_gpu_square>::en_cut(gpu::en_cut);
cuda::symbol<float> ljfluid_base<ljfluid_impl_gpu_square>::rri_smooth(gpu::rri_smooth);

/**
 * device function wrappers
 */
cuda::function<void (float3*, const float2)>
    ljfluid_base<ljfluid_impl_gpu_square>::sample_smooth_function(gpu::sample_smooth_function);

cuda::function<void (float4*, float4*, float4*, float4 const*)>
    ljfluid<ljfluid_impl_gpu_square<3> >::inteq(gpu::inteq<float3>);
cuda::function<void (float4*, float4*, float4*, float*, float*)>
    ljfluid<ljfluid_impl_gpu_square<3> >::mdstep(gpu::mdstep<float3, dfloat3>);

cuda::function<void (float2*, float2*, float2*, float2 const*)>
    ljfluid<ljfluid_impl_gpu_square<2> >::inteq(gpu::inteq<float2>);
cuda::function<void (float2*, float2*, float2*, float*, float*)>
    ljfluid<ljfluid_impl_gpu_square<2> >::mdstep(gpu::mdstep<float2, dfloat2>);

}} // namespace ljgpu::gpu
