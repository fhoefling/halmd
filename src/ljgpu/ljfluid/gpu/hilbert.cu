#include "hip/hip_runtime.h"
/* Hilbert spacing-filling curve kernel
 *
 * Copyright © 2008-2009  Peter Colberg
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <float.h>
#include <ljgpu/algorithm/gpu/base.cuh>
#include <ljgpu/math/gpu/vector2d.cuh>
#include <ljgpu/math/gpu/vector3d.cuh>
#include <ljgpu/ljfluid/gpu/hilbert.hpp>
using namespace ljgpu::gpu::hilbert;

namespace ljgpu { namespace cu { namespace hilbert
{

/** periodic box length */
__constant__ float box;
/** Hilbert space-filling curve recursion depth */
__constant__ unsigned int depth;

/**
 * swap Hilbert spacing-filling curve vertices
 */
__device__ void vertex_swap(uint& v, uint& a, uint& b, uint const& mask)
{
    // swap bits comprising Hilbert codes in vertex-to-code lookup table
    const uint va = ((v >> a) & mask);
    const uint vb = ((v >> b) & mask);
    v = v ^ (va << a) ^ (vb << b) ^ (va << b) ^ (vb << a);
    // update code-to-vertex lookup table
    swap(a, b);
}

/**
 * map 3-dimensional point to 1-dimensional point on Hilbert space curve
 */
__global__ void hilbert_curve(float4 const* g_r, unsigned int* g_sfc)
{
    //
    // Jun Wang & Jie Shan, Space-Filling Curve Based Point Clouds Index,
    // GeoComputation, 2005
    //

    // Hilbert code for particle
    unsigned int hcode = 0;

    //
    // We need to avoid ambiguities during the assignment of a particle
    // to a subcell, i.e. the particle position should never lie on an
    // edge or corner of multiple subcells, or the algorithm will have
    // trouble converging to a definite Hilbert curve.
    //
    // Therefore, we use a simple cubic lattice of predefined dimensions
    // according to the number of cells at the deepest recursion level,
    // and round the particle position to the nearest center of a cell.
    //

    // Hilbert cells per dimension at deepest recursion level
    const uint n = 1UL << depth;
    // fractional index of particle's Hilbert cell in [0, n)
    const float3 cell = (__saturatef(unpack(g_r[GTID]) / box) * (1.f - FLT_EPSILON)) * n;

    // round particle position to center of cell in unit coordinates
    float3 r = (floorf(cell) + make_float3(0.5f, 0.5f, 0.5f)) / n;
    // use symmetric coordinates
    r -= make_float3(0.5f, 0.5f, 0.5f);

    // Hilbert code-to-vertex lookup table
    uint a = 21;
    uint b = 18;
    uint c = 12;
    uint d = 15;
    uint e = 3;
    uint f = 0;
    uint g = 6;
    uint h = 9;
    // Hilbert vertex-to-code lookup table
    uint vc = 1U << b ^ 2U << c ^ 3U << d ^ 4U << e ^ 5U << f ^ 6U << g ^ 7U << h;

#define MASK ((1 << 3) - 1)

    // 32-bit integer for 3D Hilbert code allows a maximum of 10 levels
    for (unsigned int i = 0; i < depth; ++i) {
	// determine Hilbert vertex closest to particle
	const uint x = __signbitf(r.x) & 1;
	const uint y = __signbitf(r.y) & 1;
	const uint z = __signbitf(r.z) & 1;
	// lookup Hilbert code
	const uint v = (vc >> (3 * (x + (y << 1) + (z << 2))) & MASK);

	// scale particle coordinates to subcell
	r = 2 * r - make_float3(0.5f - x, 0.5f - y, 0.5f - z);
	// apply permutation rule according to Hilbert code
	if (v == 0) {
	    vertex_swap(vc, b, h, MASK);
	    vertex_swap(vc, c, e, MASK);
	}
	else if (v == 1 || v == 2) {
	    vertex_swap(vc, c, g, MASK);
	    vertex_swap(vc, d, h, MASK);
	}
	else if (v == 3 || v == 4) {
	    vertex_swap(vc, a, c, MASK);
#ifdef USE_ALTERNATIVE_HILBERT_3D
	    vertex_swap(vc, b, d, MASK);
	    vertex_swap(vc, e, g, MASK);
#endif
	    vertex_swap(vc, f, h, MASK);
	}
	else if (v == 5 || v == 6) {
	    vertex_swap(vc, a, e, MASK);
	    vertex_swap(vc, b, f, MASK);
	}
	else if (v == 7) {
	    vertex_swap(vc, a, g, MASK);
	    vertex_swap(vc, d, f, MASK);
	}

	// add vertex code to partial Hilbert code
	hcode = (hcode << 3) + v;
    }

#undef MASK

    // store Hilbert code for particle
    g_sfc[GTID] = hcode;
}

__global__ void hilbert_curve(float2 const* g_r, unsigned int* g_sfc)
{
    // Hilbert code for particle
    unsigned int hcode = 0;
    // Hilbert cells per dimension at deepest recursion level
    const uint n = 1UL << depth;
    // fractional index of particle's Hilbert cell in [0, n)
    const float2 cell = (__saturatef(unpack(g_r[GTID]) / box) * (1.f - FLT_EPSILON)) * n;

    // round particle position to center of cell in unit coordinates
    float2 r = (floorf(cell) + make_float2(0.5f, 0.5f)) / n;
    // use symmetric coordinates
    r -= make_float2(0.5f, 0.5f);

    // Hilbert code-to-vertex lookup table
    uint a = 6;
    uint b = 4;
    uint c = 0;
    uint d = 2;
    // Hilbert vertex-to-code lookup table
    uint vc = 1U << b ^ 2U << c ^ 3U << d;

#define MASK ((1 << 2) - 1)

    // 32-bit integer for 2D Hilbert code allows a maximum of 16 levels
    for (unsigned int i = 0; i < depth; ++i) {
	// determine Hilbert vertex closest to particle
	const uint x = __signbitf(r.x) & 1;
	const uint y = __signbitf(r.y) & 1;
	// lookup Hilbert code
	const uint v = (vc >> (2 * (x + (y << 1))) & MASK);

	// scale particle coordinates to subcell
	r = 2 * r - make_float2(0.5f - x, 0.5f - y);
	// apply permutation rule according to Hilbert code
	if (v == 0) {
	    vertex_swap(vc, b, d, MASK);
	}
	else if (v == 3) {
	    vertex_swap(vc, a, c, MASK);
	}

	// add vertex code to partial Hilbert code
	hcode = (hcode << 2) + v;
    }

#undef MASK

    // store Hilbert code for particle
    g_sfc[GTID] = hcode;
}

}}} // namespace ljgpu::cu::hilbert

namespace ljgpu { namespace gpu
{

/**
 * device function wrappers
 */
cuda::function<void (float4 const*, unsigned int*),
	       void (float2 const*, unsigned int*)>
    hilbert::curve(cu::hilbert::hilbert_curve, cu::hilbert::hilbert_curve);

/**
 * device constant wrappers
 */
cuda::symbol<float> hilbert::box(cu::hilbert::box);
cuda::symbol<unsigned int> hilbert::depth(cu::hilbert::depth);

}} // namespace ljgpu::gpu
