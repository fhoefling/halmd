#include "hip/hip_runtime.h"
/* Parallelized rand48 random number generator for CUDA
 *
 * Copyright © 2007-2009  Peter Colberg
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <ljgpu/algorithm/gpu/base.cuh>
#include <ljgpu/rng/gpu/rand48.cuh>
#include <ljgpu/rng/gpu/rand48.hpp>
using namespace ljgpu::gpu::rand48;

namespace ljgpu { namespace cu { namespace rand48
{

/*
 * This is a parallel version of the Unix rand48 generator for CUDA.
 * It is based on the GNU Scientific Library rand48 implementation.
 */

/**
 * compute leapfrog multipliers for initialization
 */
__global__ void leapfrog(uint48* g_la)
{
    const uint48 a(0xE66D, 0xDEEC, 0x0005);

    //
    // leapfrog multiplier:
    //   A = a^N mod m
    //

    uint48 A = a;
    uint48 x = a;

    // fast exponentiation by squares
    for (uint k = GTID; k > 0; k >>= 1) {
        if (k % 2 == 1) {
            A = muladd(x, A, 0);
        }
        x = muladd(x, x, 0);
    }

    g_la[GTID] = A;
}

/**
 * initialize generator with 32-bit integer seed
 */
__global__ void set(uint48 const* g_la, uint48 const* g_lc, uint48 *g_a, uint48 *g_c, uint seed)
{
    const uint48 c(0x000B, 0, 0);

    //
    // leapfrog addend:
    //   C = (c * sum(n = 0..(N-1), a^n)) mod m
    //

    const uint48 A = g_la[GTID];
    const uint48 C = muladd(c, g_lc[GTID], c);

    if (GTID == GTDIM - 1) {
        // store leapfrog constants
        *g_a = A;
        *g_c = C;
    }

    // default seed
    ushort3 x = make_ushort3(0x330E, 0xABCD, 0x1234);

    if (seed > 0) {
        x.y = seed & 0xFFFF;
        x.z = (seed >> 16) & 0xFFFF;
    }

    // generate initial state
    g_state[GTID] = muladd(A, x, C);
}

/**
 * restore generate state
 */
__global__ void restore(uint48 const* g_la, uint48 const* g_lc, uint48 *g_a, uint48 *g_c, ushort3 state)
{
    const uint48 c(0x000B, 0, 0);

    const uint48 A = g_la[GTID];
    const uint48 C = muladd(c, g_lc[GTID], c);

    if (GTID == GTDIM - 1) {
        // store leapfrog constants
        *g_a = A;
        *g_c = C;

        g_state[0] = state;
    }
    else {
        // generate initial states
        g_state[GTID + 1] = muladd(A, state, C);
    }
}

/**
 * save generator state
 */
__global__ void save(ushort3 *state)
{
    if (GTID == 0) {
        *state = g_state[0];
    }
}

/**
 * fill array with uniform random numbers in [0.0, 1.0)
 */
__global__ void uniform(float* v, uint len)
{
    ushort3 x = g_state[GTID];

    for (uint k = GTID; k < len; k += GTDIM) {
        v[k] = uniform(x);
    }

    g_state[GTID] = x;
}

/**
 * returns random integer in [0, 2^32-1]
 */
__device__ uint get(ushort3& state)
{
    uint r = (state.z << 16UL) + state.y;
    state = muladd(a, state, c);
    return r;
}

/**
 * fill array with random integers in [0, 2^32-1]
 */
__global__ void get(uint* v, uint len)
{
    ushort3 x = g_state[GTID];

    for (uint k = GTID; k < len; k += GTDIM) {
        v[k] = get(x);
    }

    g_state[GTID] = x;
}

}}} // namespace ljgpu::cu::rand48

namespace ljgpu { namespace gpu
{

/**
 * device function wrappers
 */
cuda::function<void (uint48*)>
    rand48::leapfrog(cu::rand48::leapfrog);
cuda::function<void (uint48 const*, uint48 const*, uint48*, uint48*, uint)>
    rand48::set(cu::rand48::set);
cuda::function<void (uint48 const*, uint48 const*, uint48*, uint48*, ushort3)>
    rand48::restore(cu::rand48::restore);
cuda::function<void (ushort3*)>
    rand48::save(cu::rand48::save);
cuda::function<void (float*, uint)>
    rand48::uniform(cu::rand48::uniform);
cuda::function<void (uint*, uint)>
    rand48::get(cu::rand48::get);

/**
 * device constant wrappers
 */
cuda::symbol<uint48> rand48::a(cu::rand48::a);
cuda::symbol<uint48> rand48::c(cu::rand48::c);
cuda::symbol<ushort3*> rand48::state(cu::rand48::g_state);

}} // namespace ljgpu::gpu
