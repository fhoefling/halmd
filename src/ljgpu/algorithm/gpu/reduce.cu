#include "hip/hip_runtime.h"
/* Parallel reduction kernel
 *
 * Copyright © 2008-2009  Peter Colberg
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <ljgpu/algorithm/gpu/base.cuh>
#include <ljgpu/algorithm/gpu/reduce.hpp>
#include <ljgpu/math/gpu/vector2d.cuh>
#include <ljgpu/math/gpu/vector3d.cuh>

namespace ljgpu { namespace cu { namespace reduce
{

enum { THREADS = gpu::reduce::THREADS };

/**
 * unary transformations
 */
template <typename T, typename U>
__device__ T identity_(U v)
{
    return v;
}

template <>
__device__ float3 identity_(float4 v)
{
    return make_float3(v.x, v.y, v.z);
}

template <>
__device__ float4 identity_(float3 v)
{
    return make_float4(v.x, v.y, v.z, 0);
}

template <typename T, typename U>
__device__ T square_(U v)
{
    return v * v;
}

/**
 * binary transformations
 */
template <typename T>
__device__ T sum_(T v1, T v2)
{
    return v1 + v2;
}

/**
 * parallel reduction
 */
template <typename input_type, typename output_type,
	  output_type (*input_function)(input_type),
	  output_type (*reduce_function)(output_type, output_type),
	  output_type (*output_function)(output_type),
	  typename T, typename U>
__device__ void reduce(T const* g_in, U* g_block_sum, uint n)
{
    __shared__ output_type s_vv[THREADS];

    // load values from global device memory
    output_type vv = 0;
    for (uint i = GTID; i < n; i += GTDIM) {
	input_type v = identity_<input_type>(g_in[i]);
	vv = reduce_function(vv, input_function(v));
    }
    // reduced value for this thread
    s_vv[TID] = vv;
    __syncthreads();

    // compute reduced value for all threads in block
    if (TID < 256) {
	vv = reduce_function(vv, s_vv[TID + 256]);
	s_vv[TID] = vv;
    }
    __syncthreads();
    if (TID < 128) {
	vv = reduce_function(vv, s_vv[TID + 128]);
	s_vv[TID] = vv;
    }
    __syncthreads();
    if (TID < 64) {
	vv = reduce_function(vv, s_vv[TID + 64]);
	s_vv[TID] = vv;
    }
    __syncthreads();
    if (TID < 32) {
	vv = reduce_function(vv, s_vv[TID + 32]);
	s_vv[TID] = vv;
    }
    // no further syncs needed within execution warp of 32 threads
    if (TID < 16) {
	vv = reduce_function(vv, s_vv[TID + 16]);
	s_vv[TID] = vv;
    }
    if (TID < 8) {
	vv = reduce_function(vv, s_vv[TID + 8]);
	s_vv[TID] = vv;
    }
    if (TID < 4) {
	vv = reduce_function(vv, s_vv[TID + 4]);
	s_vv[TID] = vv;
    }
    if (TID < 2) {
	vv = reduce_function(vv, s_vv[TID + 2]);
	s_vv[TID] = vv;
    }
    if (TID < 1) {
	vv = reduce_function(vv, s_vv[TID + 1]);
	// store block reduced value in global memory
	g_block_sum[blockIdx.x] = identity_<U>(output_function(vv));
    }
}

/**
 * blockwise sum
 */
template <typename input_type, typename output_type, typename T, typename U>
__global__ void sum(T const* g_in, U* g_block_sum, uint n)
{
    reduce<input_type, output_type, identity_, sum_, identity_>(g_in, g_block_sum, n);
}

/**
 * blockwise absolute maximum
 */
template <typename input_type, typename output_type, typename T, typename U>
__global__ void max(T const* g_in, U* g_block_max, uint n)
{
    reduce<input_type, output_type, square_, fmaxf, sqrtf>(g_in, g_block_max, n);
}

}}} // namespace ljgpu::cu::reduce

namespace ljgpu { namespace gpu
{

/**
 * device function wrappers
 */
cuda::function<void(float const*, dfloat*, uint)>
	       reduce::sum(cu::reduce::sum<float, dfloat>);
cuda::function<void(float4 const*, float*, uint),
	       void(float2 const*, float*, uint)>
	       reduce::max(cu::reduce::max<float3, float>,
			   cu::reduce::max<float2, float>);

}} // namespace ljgpu::gpu
