#include "hip/hip_runtime.h"
/*
 * Copyright © 2008-2009  Peter Colberg
 *
 * This file is part of HALMD.
 *
 * HALMD is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <halmd/algorithm/gpu/radix_sort_kernel.hpp>
#include <halmd/algorithm/gpu/scan_kernel.cuh>

namespace halmd
{
namespace algorithm { namespace gpu
{
namespace radix_sort_kernel
{

/**
 * atomically add value to 32-bit word in shared memory
 */
template <uint count>
__device__ void atomic_add(uint const& i, uint const& value, uint& r)
{
    extern __shared__ uint s_bucket[];
    const uint tid = threadIdx.x;

    // increment shared memory address within single thread of each half-warp
    if ((tid % HALF_WARP_SIZE) == (HALF_WARP_SIZE - count)) {
        r = s_bucket[i];
        s_bucket[i] = r + value;
    }
    __syncthreads();

    // recurse through all threads of each half-warp
    atomic_add<count - 1>(i, value, r);
}

template <> __device__ void atomic_add<0>(uint const&, uint const&, uint&) {}

/**
 * returns 32-bit word in shared memory and atomically adds value
 */
__device__ uint atomic_add(uint const& i, uint const& value)
{
    uint r;
    atomic_add<HALF_WARP_SIZE>(i, value, r);
    return r;
}

/**
 * compute partial radix counts for given radix shift
 */
__global__ void histogram_keys(uint const* g_in, uint* g_bucket, const uint count, const uint shift)
{
    //
    // Radix Sort for Vector Multiprocessors,
    // Marco Zagha and Guy E. Blelloch.
    // Supercomputing '91, November 1991. 
    //
    // http://www.cs.cmu.edu/~scandal/papers/cray-sort-supercomputing91.html
    //

    extern __shared__ uint s_bucket[];

    const uint tid = threadIdx.x;
    const uint pid = threadIdx.x / HALF_WARP_SIZE;
    const uint wid = threadIdx.x % HALF_WARP_SIZE;
    const uint threads = blockDim.x;
    const uint bid = blockIdx.x;
    const uint blocks = gridDim.x;

    // set bucket counts to zero
    for (uint i = 0; i < BUCKETS_PER_THREAD; ++i) {
        s_bucket[tid + i * threads] = 0;
    }
    __syncthreads();

    // number of partitions per block
    const uint parts = threads / HALF_WARP_SIZE;
    // number of elements per partition, aligned to total thread count
    const uint elems = ((count + blocks * threads - 1) / (blocks * threads)) * HALF_WARP_SIZE;

    for (uint i = 0; i < elems; i += HALF_WARP_SIZE) {
        // global memory offset of sort key
        const uint j = i + wid + (pid + parts * bid) * elems;
        // read sort key and transform according to radix shift
        const uint radix = (j < count) ? (g_in[j] >> shift) & (BUCKET_SIZE - 1) : 0;

        // atomically increment bucket count
        atomic_add(pid + parts * radix, (j < count) ? 1 : 0);
    }

    // write radix counts to global memory
    for (uint i = 0; i < BUCKETS_PER_THREAD; ++i) {
        // partition
        const uint j = tid % parts;
        // bucket
        const uint k = tid / parts + i * threads / parts;
        // write count to partition bucket in column major order
        g_bucket[j + (bid + k * blocks) * parts] = s_bucket[tid + i * threads];
    }
}

/**
 * permute array given radix counts prefix sums
 */
template <typename T>
__global__ void permute(uint const* g_in, uint* g_out, T const* g_data_in, T* g_data_out, uint const* g_bucket, const uint count, const uint shift)
{
    extern __shared__ uint s_bucket[];

    const uint tid = threadIdx.x;
    const uint pid = threadIdx.x / HALF_WARP_SIZE;
    const uint wid = threadIdx.x % HALF_WARP_SIZE;
    const uint threads = blockDim.x;
    const uint bid = blockIdx.x;
    const uint blocks = gridDim.x;

    // number of partitions per block
    const uint parts = threads / HALF_WARP_SIZE;
    // number of elements per partition, aligned to total thread count
    const uint elems = ((count + blocks * threads - 1) / (blocks * threads)) * HALF_WARP_SIZE;

    // read radix counts from global memory
    for (uint i = 0; i < BUCKETS_PER_THREAD; ++i) {
        // partition
        const uint j = tid % parts;
        // bucket
        const uint k = tid / parts + i * threads / parts;
        // read count from partition bucket in column major order
        s_bucket[tid + i * threads] = g_bucket[j + (bid + k * blocks) * parts];
    }
    __syncthreads();

    for (uint i = 0; i < elems; i += HALF_WARP_SIZE) {
        // global memory offset of sort key
        const uint j = i + wid + (pid + parts * bid) * elems;
        // read sort key from global memory
        const uint key = (j < count) ? g_in[j] : 0;
        // transform sort key according to radix shift
        const uint radix = (key >> shift) & (BUCKET_SIZE - 1);

        // atomically read and increment global radix offset
        const uint l = atomic_add(pid + parts * radix, (j < count) ? 1 : 0);

        // scatter write permuted array element to global memory
        if (j < count) {
            // write sort key
            g_out[l] = key;
            // permute data array element
            g_data_out[l] = g_data_in[j];
        }
    }
}

} // namespace radix_sort_kernel

/**
 * device function wrappers
 */
template <typename T>
radix_sort_wrapper<T> const radix_sort_wrapper<T>::kernel = {
    radix_sort_kernel::histogram_keys
  , radix_sort_kernel::permute
};

template class radix_sort_wrapper<int>;
template class radix_sort_wrapper<unsigned int>;
template class radix_sort_wrapper<float4>;
template class radix_sort_wrapper<float2>;

}} // namespace algorithm::gpu

template class algorithm::gpu::scan_wrapper<unsigned int>;

} // namespace halmd
