#include "hip/hip_runtime.h"
/* Parallel exclusive prefix sum
 *
 * Copyright © 2008-2009  Peter Colberg
 *
 * This file is part of HALMD.
 *
 * HALMD is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <halmd/algorithm/gpu/base.cuh>
#include <halmd/algorithm/gpu/prefix_sum.hpp>

using namespace halmd::gpu::prefix_sum;

namespace halmd { namespace cu { namespace prefix_sum
{

/**
 * blockwise parallel exclusive prefix sum
 */
template <typename T>
__device__ T grid_prefix_sum(T const* g_in, T* g_out, const uint count)
{
    //
    // Prefix Sums and Their Applications,
    // Guy E. Blelloch.
    // CMU-CS-90-190, November 1990.
    //
    // http://www.cs.cmu.edu/~scandal/papers/CMU-CS-90-190.html
    //

    //
    // Parallel Prefix Sum (Scan) with CUDA,
    // Mark Harris, April 2007, NVIDIA Corporation
    //

    extern __shared__ T s_array[];
    T block_sum = 0;

    const uint tid = threadIdx.x;
    const uint threads = blockDim.x;
    const uint bid = blockIdx.x;

    // read elements from global memory, or pad with zero
    const uint i1 = 2 * bid * threads + tid;
    const uint i2 = (2 * bid + 1) * threads + tid;
    s_array[boff(tid)] = (i1 < count) ? g_in[i1] : 0;
    s_array[boff(threads + tid)] = (i2 < count) ? g_in[i2] : 0;
    __syncthreads();

    // up-sweep phase from leaves to root of binary tree
    for (uint d = threads, n = 1; d > 0; d >>= 1, n <<= 1) {
        if (tid < d) {
            s_array[boff(n * (2 * tid + 2) - 1)] += s_array[boff(n * (2 * tid + 1) - 1)];
        }
        __syncthreads();
    }

    if (tid == 0) {
        // set last element to zero for down-sweep phase
        swap(s_array[boff(2 * threads - 1)], block_sum);
    }
    __syncthreads();

    // down-sweep phase from root to leaves of binary tree
    for (uint d = 1, n = threads; n > 0; d <<= 1, n >>= 1) {
        if (tid < d) {
            const uint i1 = boff(n * (2 * tid + 1) - 1);
            const uint i2 = boff(n * (2 * tid + 2) - 1);
            const T t1 = s_array[i1];
            const T t2 = s_array[i2];
            s_array[i1] = t2;
            s_array[i2] = t1 + t2;
        }
        __syncthreads();
    }

    // write partial prefix sums to global memory
    if (i1 < count)
        g_out[i1] = s_array[boff(tid)];
    if (i2 < count)
        g_out[i2] = s_array[boff(threads + tid)];

    // block sum for last thread in block, otherwise zero
    return block_sum;
}

/**
 * blockwise parallel exclusive prefix sum
 */
template <typename T>
__global__ void grid_prefix_sum(T const* g_in, T* g_out, T* g_block_sum, const uint count)
{
    const uint tid = threadIdx.x;
    const uint bid = blockIdx.x;

    const T block_sum =  grid_prefix_sum(g_in, g_out, count);

    if (tid == 0) {
        g_block_sum[bid] = block_sum;
    }
}

/**
 * single-block parallel exclusive prefix sum
 */
template <typename T>
__global__ void block_prefix_sum(T const* g_in, T* g_out, const uint count)
{
    grid_prefix_sum(g_in, g_out, count);
}

/**
 * add block prefix sum to partial prefix sums for each block
 */
template <typename T>
__global__ void add_block_sums(T const* g_in, T* g_out, T const* g_block_sum, const uint count)
{
    __shared__ T s_block_sum[1];

    const uint tid = threadIdx.x;
    const uint threads = blockDim.x;
    const uint bid = blockIdx.x;

    if (tid == 0) {
        // read block sum for subsequent shared memory broadcast
        s_block_sum[0] = g_block_sum[bid];
    }
    __syncthreads();

    const uint i1 = 2 * bid * threads + tid;
    if (i1 < count)
        g_out[i1] = g_in[i1] + s_block_sum[0];

    const uint i2 = (2 * bid + 1) * threads + tid;
    if (i2 < count)
        g_out[i2] = g_in[i2] + s_block_sum[0];
}

}}} // namespace halmd::cu::prefix_sum

namespace halmd { namespace gpu
{

/**
 * device function wrappers
 */
cuda::function<void (uint const*, uint*, uint*, const uint),
               void (uint48 const*, uint48*, uint48*, const uint)>
               prefix_sum::grid_prefix_sum(cu::prefix_sum::grid_prefix_sum,
                                           cu::prefix_sum::grid_prefix_sum);
cuda::function<void (uint const*, uint*, uint const*, const uint),
               void (uint48 const*, uint48*, uint48 const*, const uint)>
               prefix_sum::add_block_sums(cu::prefix_sum::add_block_sums,
                                          cu::prefix_sum::add_block_sums);
cuda::function<void (uint48 const*, uint48*, const uint),
               void (uint const*, uint*, const uint)>
               prefix_sum::block_prefix_sum(cu::prefix_sum::block_prefix_sum,
                                            cu::prefix_sum::block_prefix_sum);

}} // namespace halmd::gpu
