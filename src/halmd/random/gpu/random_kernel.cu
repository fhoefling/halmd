#include "hip/hip_runtime.h"
/*
 * Copyright © 2010  Peter Colberg
 *
 * This file is part of HALMD.
 *
 * HALMD is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <halmd/algorithm/gpu/scan_kernel.cuh>
#include <halmd/random/gpu/normal_distribution.cuh>
#include <halmd/random/gpu/random_number_generator.cuh>
#include <halmd/random/gpu/random_kernel.hpp>
#include <halmd/utility/gpu/thread.cuh>

namespace halmd
{
namespace random { namespace gpu
{
namespace random_kernel
{

// import into current namespace
using random::gpu::rng;

/**
 * fill array with uniform random numbers in [0.0, 1.0)
 */
template <typename RandomNumberGenerator>
__global__ void uniform(float* v, unsigned int len)
{
    typename RandomNumberGenerator::state_type state = rng<rand48_rng>::g_rng[GTID];

    for (unsigned int k = GTID; k < len; k += GTDIM) {
        v[k] = uniform(rng<rand48_rng>::g_rng, state);
    }

    rng<rand48_rng>::g_rng[GTID] = state;
}

/**
 * fill array with random integers in [0, 2^32-1]
 */
template <typename RandomNumberGenerator>
__global__ void get(unsigned int* v, unsigned int len)
{
    typename RandomNumberGenerator::state_type state = rng<rand48_rng>::g_rng[GTID];

    for (unsigned int k = GTID; k < len; k += GTDIM) {
        v[k] = get(rng<rand48_rng>::g_rng, state);
    }

    rng<rand48_rng>::g_rng[GTID] = state;
}

/**
 * fill array with normal distributed random numbers in [0.0, 1.0)
 */
template <typename RandomNumberGenerator>
__global__ void normal(float* v, unsigned int len, float mean, float sigma)
{
    typename RandomNumberGenerator::state_type state = rng<rand48_rng>::g_rng[GTID];

    for (unsigned int k = GTID; k < len; k += 2 * GTDIM) {
        normal(rng<rand48_rng>::g_rng, state, v[k], v[k + GTID], mean, sigma);
    }

    rng<rand48_rng>::g_rng[GTID] = state;
}


} // namespace random_kernel

/**
 * CUDA C++ wrappers
 */
template <typename T>
random_wrapper<T> const random_wrapper<T>::kernel = {
    random_kernel::rng<T>::g_rng
  , random_kernel::uniform<T>
  , random_kernel::get<T>
  , random_kernel::normal<T>
};

template class random_wrapper<random::gpu::rand48_rng>;

}} // namespace random::gpu

} // namespace halmd
