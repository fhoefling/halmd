#include "hip/hip_runtime.h"
/*
 * Copyright © 2008-2010  Peter Colberg
 *
 * This file is part of HALMD.
 *
 * HALMD is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <boost/mpl/if.hpp>

#include <halmd/algorithm/gpu/base.cuh>
#include <halmd/mdsim/gpu/box_kernel.cuh>
#include <halmd/mdsim/gpu/force_kernel.cuh>
#include <halmd/mdsim/gpu/forces/lj_kernel.cuh>
#include <halmd/mdsim/gpu/particle_kernel.cuh>
#include <halmd/numeric/gpu/blas/dsfloat.cuh>
#include <halmd/numeric/gpu/blas/symmetric.cuh>
#include <halmd/numeric/gpu/blas/vector.cuh>

using namespace boost::mpl;
using namespace halmd::mdsim::gpu::particle_kernel;
using namespace halmd::numeric::gpu::blas;

namespace halmd { namespace mdsim { namespace gpu { namespace forces { namespace lj_kernel
{

template <size_t N>
struct dim_
{
    /** positions, tags */
    static texture<float4, 1, hipReadModeElementType> r;
    /** cubic box edgle length */
    static __constant__ typename if_c<N == 3, float3, float2>::type box_length;
};

// explicit instantiation
template class dim_<3>;
template class dim_<2>;

/** number of placeholders per neighbor list */
__constant__ unsigned int neighbor_size_;
/** neighbor list stride */
__constant__ unsigned int neighbor_stride_;
/** Lennard-Jones potential parameters */
texture<float4, 1, hipReadModeElementType> ljparam_;

/**
 * Compute Lennard-Jones forces
 */
template <typename vector_type, typename gpu_vector_type>
__global__ void compute(
  gpu_vector_type* g_f,
  unsigned int* g_neighbor,
  float* g_en_pot,
  gpu_vector_type* g_virial)
{
    enum { dimension = vector_type::static_size };
    typedef typename vector_type::value_type value_type;
    unsigned int i = GTID;

    // load particle associated with this thread
    unsigned int type1;
    vector_type r1 = untagged<vector_type>(tex1Dfetch(dim_<dimension>::r, i), type1);

    // potential energy contribution
    float en_pot_ = 0;
    // virial contribution
    vector<float, (dimension - 1) * dimension / 2 + 1> virial_ = 0;
#ifdef USE_FORCE_DSFUN
    // force sum
    vector<dsfloat, dimension> f = 0;
#else
    vector_type f = 0;
#endif

    for (unsigned int k = 0; k < neighbor_size_; ++k) {
        // coalesced read from neighbor list
        unsigned int j = g_neighbor[k * neighbor_stride_ + i];
        // skip placeholder particles
        if (j == particle_kernel::PLACEHOLDER) {
            break;
        }

        // load particle
        unsigned int type2;
        vector_type r2 = untagged<vector_type>(tex1Dfetch(dim_<dimension>::r, j), type2);
        // Lennard-Jones potential parameters
        vector<float, 4> lj = tex1Dfetch(ljparam_, symmetric_matrix::lower_index(type1, type2));

        // particle distance vector
        vector_type r = r1 - r2;
        // enforce periodic boundary conditions
        box_kernel::reduce_periodic(r, static_cast<vector_type>(dim_<dimension>::box_length));
        // squared particle distance
        value_type rr = inner_prod(r, r);
        // enforce cutoff length
        if (rr >= lj[RR_CUT]) {
            return;
        }

        // compute Lennard-Jones force in reduced units
        value_type rri = lj[SIGMA2] / rr;
        value_type ri6 = rri * rri * rri;
        value_type fval = 48 * lj[EPSILON] * rri * ri6 * (ri6 - 0.5f) / lj[SIGMA2];
        value_type en_pot = 4 * lj[EPSILON] * ri6 * (ri6 - 1) - lj[EN_CUT];

        // virial equation sum
        virial_ += 0.5f * fval * force_kernel::virial_tensor(rr, r);
        // potential energy contribution of this particle
        en_pot_ += 0.5f * en_pot;
        // force from other particle acting on this particle
        f += fval * r;
    }

    g_f[i] = static_cast<vector_type>(f);
    g_en_pot[i] = en_pot_;
    g_virial[i] = virial_;
}

}}}}} // namespace halmd::mdsim::gpu::forces::lj_kernel
