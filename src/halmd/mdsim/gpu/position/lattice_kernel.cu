#include "hip/hip_runtime.h"
/*
 * Copyright © 2008-2009  Peter Colberg
 *
 * This file is part of HALMD.
 *
 * HALMD is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <halmd/mdsim/gpu/position/lattice_kernel.cuh>
#include <halmd/mdsim/gpu/particle_kernel.cuh>
#include <halmd/numeric/gpu/blas/vector.cuh>
#include <halmd/utility/gpu/thread.cuh>

using namespace halmd::numeric::gpu::blas;
using namespace halmd::mdsim::gpu::particle_kernel;

namespace halmd { namespace mdsim { namespace gpu { namespace position
{

namespace lattice_kernel
{

/**
 * place particles on a face centered cubic lattice (fcc)
 */
__device__ void fcc(vector<float, 3>& r, uint n)
{
    // compose primitive vectors from 1-dimensional index
    r.x = ((GTID >> 2) % n) + ((GTID ^ (GTID >> 1)) & 1) / 2.f;
    r.y = ((GTID >> 2) / n % n) + (GTID & 1) / 2.f;
    r.z = ((GTID >> 2) / n / n) + (GTID & 2) / 4.f;
}

__device__ void fcc(vector<float, 2>& r, uint n)
{
    r.x = ((GTID >> 1) % n) + (GTID & 1) / 2.f;
    r.y = ((GTID >> 1) / n) + (GTID & 1) / 2.f;
}

/**
 * place particles on a simple cubic lattice (sc)
 */
__device__ void sc(vector<float, 3>& r, uint n)
{
    r.x = (GTID % n) + 0.5f;
    r.y = (GTID / n % n) + 0.5f;
    r.z = (GTID / n / n) + 0.5f;
}

__device__ void sc(vector<float, 2>& r, uint n)
{
    r.x = (GTID % n) + 0.5f;
    r.y = (GTID / n) + 0.5f;
}

template <int dimension, void (*primitive)(vector<float, dimension>&, uint)>
__global__ void lattice(float4* g_r, uint n, float box)
{
    vector<float, dimension> r;
    primitive(r, n);
    g_r[GTID] = tagged(r * (box / n), GTID);
}

} // namespace lattice_kernel

/**
 * device function wrappers
 */
template <> cuda::function <void (float4*, uint, float)>
    lattice_wrapper<3>::fcc(lattice_kernel::lattice<3, lattice_kernel::fcc>);
template <> cuda::function <void (float4*, uint, float)>
    lattice_wrapper<3>::sc(lattice_kernel::lattice<3, lattice_kernel::sc>);
template <> cuda::function <void (float4*, uint, float)>
    lattice_wrapper<2>::fcc(lattice_kernel::lattice<2, lattice_kernel::fcc>);
template <> cuda::function <void (float4*, uint, float)>
    lattice_wrapper<2>::sc(lattice_kernel::lattice<2, lattice_kernel::sc>);

}}}} // namespace halmd::mdsim::gpu::position
