#include "hip/hip_runtime.h"
/*
 * Copyright © 2010  Peter Colberg
 *
 * This file is part of HALMD.
 *
 * HALMD is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <halmd/algorithm/gpu/reduce_kernel.cuh>
#include <halmd/numeric/blas/blas.hpp>
#include <halmd/numeric/mp/dsfloat.hpp>

using namespace halmd::algorithm::gpu;

namespace halmd
{
namespace mdsim { namespace gpu
{
namespace thermodynamics_kernel
{

} // namespace thermodynamics_kernel

}} // namespace mdsim::gpu

template class reduce_wrapper<
    sum_                        // reduce_transform
  , fixed_vector<float, 3>      // input_type
  , float4                      // coalesced_input_type
  , dsfloat                     // output_type
  , dsfloat                     // coalesced_output_type
  , square_                     // input_transform
>;

template class reduce_wrapper<
    sum_                        // reduce_transform
  , fixed_vector<float, 2>      // input_type
  , float4                      // coalesced_input_type
  , dsfloat                     // output_type
  , dsfloat                     // coalesced_output_type
  , square_                     // input_transform
>;

template class reduce_wrapper<
    sum_                        // reduce_transform
  , fixed_vector<float, 3>      // input_type
  , float4                      // coalesced_input_type
  , fixed_vector<dsfloat, 3>    // output_type
  , fixed_vector<dsfloat, 3>    // coalesced_output_type
>;

template class reduce_wrapper<
    sum_                        // reduce_transform
  , fixed_vector<float, 2>      // input_type
  , float4                      // coalesced_input_type
  , fixed_vector<dsfloat, 2>    // output_type
  , fixed_vector<dsfloat, 2>    // coalesced_output_type
>;

template class reduce_wrapper<
    sum_                        // reduce_transform
  , float                       // input_type
  , float                       // coalesced_input_type
  , dsfloat                     // output_type
  , dsfloat                     // coalesced_output_type
>;

} // namespace halmd
