#include "hip/hip_runtime.h"
/*
 * Copyright © 2008-2010  Peter Colberg
 *
 * This file is part of HALMD.
 *
 * HALMD is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <halmd/mdsim/gpu/neighbour_kernel.cu>
#include <halmd/mdsim/gpu/neighbour_wrapper.cuh>

namespace halmd { namespace mdsim { namespace gpu
{

cuda::texture<float>
  neighbour_wrapper<3>::rr_cut_skin = neighbour_kernel::rr_cut_skin_;
cuda::symbol<uint3>
  neighbour_wrapper<3>::ncell = neighbour_kernel::dim_<3>::ncell;
cuda::symbol<unsigned int>
  neighbour_wrapper<3>::neighbour_size = neighbour_kernel::neighbour_size_;
cuda::symbol<unsigned int>
  neighbour_wrapper<3>::neighbour_stride = neighbour_kernel::neighbour_stride_;
cuda::symbol<unsigned int>
  neighbour_wrapper<3>::nbox = neighbour_kernel::nbox_;
cuda::texture<float4>
  neighbour_wrapper<3>::r = neighbour_kernel::dim_<3>::r;
cuda::symbol<float3>
  neighbour_wrapper<3>::box_length = neighbour_kernel::dim_<3>::box_length;
cuda::symbol<float3>
  neighbour_wrapper<3>::cell_length = neighbour_kernel::dim_<3>::cell_length;
cuda::function<void (unsigned int*, unsigned int const*, unsigned int const*, unsigned int const*, unsigned int*)>
  neighbour_wrapper<3>::assign_cells = neighbour_kernel::assign_cells;
cuda::function<void (unsigned int*, unsigned int*)>
  neighbour_wrapper<3>::find_cell_offset = neighbour_kernel::find_cell_offset;
cuda::function<void (unsigned int*)>
  neighbour_wrapper<3>::gen_index = neighbour_kernel::gen_index;
cuda::function<void (unsigned int*, unsigned int*, unsigned int const*)>
  neighbour_wrapper<3>::update_neighbours = neighbour_kernel::update_neighbours<3>;
cuda::function<void (float4 const*, unsigned int*)>
  neighbour_wrapper<3>::compute_cell = neighbour_kernel::compute_cell<3>;

cuda::texture<float>
  neighbour_wrapper<2>::rr_cut_skin = neighbour_kernel::rr_cut_skin_;
cuda::symbol<uint2>
  neighbour_wrapper<2>::ncell = neighbour_kernel::dim_<2>::ncell;
cuda::symbol<unsigned int>
  neighbour_wrapper<2>::neighbour_size = neighbour_kernel::neighbour_size_;
cuda::symbol<unsigned int>
  neighbour_wrapper<2>::neighbour_stride = neighbour_kernel::neighbour_stride_;
cuda::symbol<unsigned int>
  neighbour_wrapper<2>::nbox = neighbour_kernel::nbox_;
cuda::texture<float4>
  neighbour_wrapper<2>::r = neighbour_kernel::dim_<2>::r;
cuda::symbol<float2>
  neighbour_wrapper<2>::box_length = neighbour_kernel::dim_<2>::box_length;
cuda::symbol<float2>
  neighbour_wrapper<2>::cell_length = neighbour_kernel::dim_<2>::cell_length;
cuda::function<void (unsigned int*, unsigned int const*, unsigned int const*, unsigned int const*, unsigned int*)>
  neighbour_wrapper<2>::assign_cells = neighbour_kernel::assign_cells;
cuda::function<void (unsigned int*, unsigned int*)>
  neighbour_wrapper<2>::find_cell_offset = neighbour_kernel::find_cell_offset;
cuda::function<void (unsigned int*)>
  neighbour_wrapper<2>::gen_index = neighbour_kernel::gen_index;
cuda::function<void (unsigned int*, unsigned int*, unsigned int const*)>
  neighbour_wrapper<2>::update_neighbours = neighbour_kernel::update_neighbours<2>;
cuda::function<void (float4 const*, unsigned int*)>
  neighbour_wrapper<2>::compute_cell = neighbour_kernel::compute_cell<2>;

}}} // namespace halmd::mdsim::gpu
