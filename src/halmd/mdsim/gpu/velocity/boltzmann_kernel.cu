#include "hip/hip_runtime.h"
/*
 * Copyright © 2008-2010  Peter Colberg and Felix Höfling
 *
 * This file is part of HALMD.
 *
 * HALMD is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <halmd/algorithm/gpu/reduction.cuh>
#include <halmd/mdsim/gpu/particle_kernel.cuh>
#include <halmd/mdsim/gpu/velocity/boltzmann_kernel.cuh>
#include <halmd/numeric/gpu/blas/dsfloat.cuh>
#include <halmd/numeric/gpu/blas/vector.cuh>
#include <halmd/utility/gpu/thread.cuh>

using namespace halmd::numeric::gpu::blas;
using namespace halmd::mdsim::gpu::particle_kernel;

//
// Maxwell-Boltzmann distribution at accurate temperature
//

namespace halmd
{
namespace mdsim { namespace gpu { namespace velocity
{
namespace boltzmann_kernel
{

enum { BLOCKS = boltzmann_wrapper<>::BLOCKS };
enum { THREADS = boltzmann_wrapper<>::THREADS };

/**
 * generate Maxwell-Boltzmann distributed velocities and reduce velocity
 */
template <typename vector_type, typename T>
__global__ void gaussian(float4* g_v, uint npart, uint nplace, float temp, T* g_vcm)
{
    enum { dimension = vector_type::static_size };
    __shared__ vector_type s_vcm[THREADS];
    vector_type vcm = 0;

//     // read random number generator state from global device memory
//     rand48::state_type state = rand48::g_state[GTID];
//
//     for (uint i = GTID; i < npart; i += GTDIM) {
//         T v;
//         rand48::gaussian(v, temp, state);
//         g_v[i] = v;
// #ifdef USE_VERLET_DSFUN
//         g_v[i + nplace] = vector<float, dimension>(0);
// #endif
//         vcm += vector<float, dimension>(v);
//     }
//     // store random number generator state in global device memory
//     rand48::g_state[GTID] = state;

    // reduced value for this thread
    s_vcm[TID] = vcm;
    __syncthreads();

    // compute reduced value for all threads in block
//     reduce<THREADS / 2, sum_>(vcm, s_vcm);

    if (TID < 1) {
        // store block reduced value in global memory
        g_vcm[blockIdx.x] = static_cast<vector<float, dimension> >(vcm);
#ifdef USE_VERLET_DSFUN
        g_vcm[blockIdx.x + BDIM] = dsfloat_lo(vcm);
#endif
    }
}

/**
 * set center of mass velocity to zero and reduce squared velocity
 */
template <typename vector_type, typename T>
__global__ void shift_velocity(float4* g_v, uint npart, uint nplace, T const* g_vcm, dsfloat* g_vv)
{
    enum { dimension = vector_type::static_size };
    __shared__ vector_type s_vcm[BLOCKS];
    __shared__ dsfloat s_vv[THREADS];
    vector_type vcm = 0;
    dsfloat vv = 0;

    // compute mean center of mass velocity from block reduced values
    for (uint i = TID; i < BLOCKS; i += TDIM) {
#ifdef USE_VERLET_DSFUN
        s_vcm[i] = vector_type(g_vcm[i], g_vcm[i + BDIM]);
#else
        s_vcm[i] = g_vcm[i];
#endif
    }
    __syncthreads();
    for (uint i = 0; i < BLOCKS; ++i) {
        vcm += s_vcm[i];
    }
    vcm /= npart;

    for (uint i = GTID; i < npart; i += GTDIM) {
#ifdef USE_VERLET_DSFUN
        vector_type v; // FIXME (g_v[i], g_v[i + nplace]);
#else
        vector_type v = g_v[i];
#endif
        v -= vcm;
        g_v[i] = tagged(static_cast<vector<float, dimension> >(v), /* FIXME */0);
#ifdef USE_VERLET_DSFUN
        g_v[i + nplace] = tagged(dsfloat_lo(v), /* FIXME */0);
#endif
        vv += inner_prod(v, v);
    }
    // reduced value for this thread
    s_vv[TID] = vv;
    __syncthreads();

    // compute reduced value for all threads in block
//     reduce<THREADS / 2, sum_>(vv, s_vv);

    if (TID < 1) {
        // store block reduced value in global memory
        g_vv[blockIdx.x] = vv;
    }
}

/**
 * rescale velocities to accurate temperature
 */
template <typename vector_type>
__global__ void scale_velocity(float4* g_v, uint npart, uint nplace, dsfloat const* g_vv, dsfloat temp)
{
    enum { dimension = vector_type::static_size };
    typedef typename vector_type::value_type float_type;
    __shared__ dsfloat s_vv[THREADS];
    dsfloat vv = 0;

    // compute squared velocity sum from block reduced values
    for (uint i = TID; i < BLOCKS; i += TDIM) {
        s_vv[i] = g_vv[i];
    }
    __syncthreads();
    for (uint i = 0; i < BLOCKS; ++i) {
        vv += s_vv[i];
    }

    float_type coeff = sqrt(temp * static_cast<int>(dimension) * (static_cast<float_type>(npart) / vv));

    for (uint i = GTID; i < npart; i += GTDIM) {
        vector_type v;
        unsigned int tag;
#ifdef USE_VERLET_DSFUN
        tie(v, tag) = untagged<vector_type>(g_v[i], g_v[i + nplace]);
#else
        tie(v, tag) = untagged<vector_type>(g_v[i]);
#endif
        v *= coeff;
#ifdef USE_VERLET_DSFUN
        tie(g_v[i], g_v[i + nplace]) = tagged(v, tag);
#else
        g_v[i] = tagged(v, tag);
#endif
    }
}

} // namespace boltzmann_kernel

#ifdef USE_VERLET_DSFUN
typedef dsfloat float_type;
#else
typedef float float_type;
#endif

/**
 * device symbol wrappers
 */
// cuda::symbol<uint48>
//     boltzmann_wrapper<>::rand48::a = rng::gpu::rand48_kernel::a;
// cuda::symbol<uint48>
//     boltzmann_wrapper<>::rand48::c = rng::gpu::rand48_kernel::c;
// cuda::symbol<ushort3*>
//     boltzmann_wrapper<>::rand48::state = rng::gpu::rand48_kernel::g_state;

/**
 * device function wrappers
 */

cuda::function<void (float4*, uint, uint, float, float4*)>
    boltzmann_wrapper<3>::gaussian(boltzmann_kernel::gaussian<vector<float_type, 3> >);
cuda::function<void (float4*, uint, uint, float4 const*, dsfloat*)>
    boltzmann_wrapper<3>::shift_velocity(boltzmann_kernel::shift_velocity<vector<float_type, 3> >);
cuda::function<void (float4*, uint, uint, dsfloat const*, dsfloat)>
    boltzmann_wrapper<3>::scale_velocity(boltzmann_kernel::scale_velocity<vector<float_type, 3> >);

cuda::function<void (float4*, uint, uint, float, float2*)>
    boltzmann_wrapper<2>::gaussian(boltzmann_kernel::gaussian<vector<float_type, 2> >);
cuda::function<void (float4*, uint, uint, float2 const*, dsfloat*)>
    boltzmann_wrapper<2>::shift_velocity(boltzmann_kernel::shift_velocity<vector<float_type, 2> >);
cuda::function<void (float4*, uint, uint, dsfloat const*, dsfloat)>
    boltzmann_wrapper<2>::scale_velocity(boltzmann_kernel::scale_velocity<vector<float_type, 2> >);

}}} // namespace mdsim::gpu::velocity

} // namespace halmd
