#include "hip/hip_runtime.h"
/* Maxwell-Boltzmann distribution at accurate temperature
 *
 * Copyright © 2008-2009  Peter Colberg
 *
 * This file is part of HALMD.
 *
 * HALMD is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <halmd/algorithm/gpu/base.cuh>
#include <halmd/algorithm/gpu/reduce.cuh>
#include <halmd/math/gpu/dsvector.cuh>
#include <halmd/math/gpu/vector2d.cuh>
#include <halmd/math/gpu/vector3d.cuh>
#include <halmd/mdsim/backend/gpu/boltzmann.hpp>
#define CU_NAMESPACE boltzmann
#include <halmd/rng/gpu/rand48.cuh>

namespace halmd { namespace cu { namespace boltzmann
{

enum { BLOCKS = halmd::gpu::boltzmann<>::BLOCKS };
enum { THREADS = halmd::gpu::boltzmann<>::THREADS };

/**
 * generate Maxwell-Boltzmann distributed velocities and reduce velocity
 */
template <typename vector_type, typename T>
__global__ void gaussian(T* g_v, uint npart, uint nplace, float temp, T* g_vcm)
{
    enum { dimension = vector_type::static_size };
    __shared__ vector_type s_vcm[THREADS];
    vector_type vcm = 0;

    // read random number generator state from global device memory
    rand48::state_type state = rand48::g_state[GTID];

    for (uint i = GTID; i < npart; i += GTDIM) {
        T v;
        rand48::gaussian(v, temp, state);
        g_v[i] = v;
#ifdef USE_VERLET_DSFUN
        g_v[i + nplace] = vector<float, dimension>(0);
#endif
        vcm += vector<float, dimension>(v);
    }
    // store random number generator state in global device memory
    rand48::g_state[GTID] = state;

    // reduced value for this thread
    s_vcm[TID] = vcm;
    __syncthreads();

    // compute reduced value for all threads in block
    reduce<THREADS / 2, sum_>(vcm, s_vcm);

    if (TID < 1) {
        // store block reduced value in global memory
        g_vcm[blockIdx.x] = static_cast<vector<float, dimension> >(vcm);
#ifdef USE_VERLET_DSFUN
        g_vcm[blockIdx.x + BDIM] = dsfloat2lo(vcm);
#endif
    }
}

/**
 * set center of mass velocity to zero and reduce squared velocity
 */
template <typename vector_type, typename T>
__global__ void shift_velocity(T* g_v, uint npart, uint nplace, T const* g_vcm, dsfloat* g_vv)
{
    enum { dimension = vector_type::static_size };
    __shared__ vector_type s_vcm[BLOCKS];
    __shared__ dsfloat s_vv[THREADS];
    vector_type vcm = 0;
    dsfloat vv = 0;

    // compute mean center of mass velocity from block reduced values
    for (uint i = TID; i < BLOCKS; i += TDIM) {
#ifdef USE_VERLET_DSFUN
        s_vcm[i] = vector_type(g_vcm[i], g_vcm[i + BDIM]);
#else
        s_vcm[i] = g_vcm[i];
#endif
    }
    __syncthreads();
    for (uint i = 0; i < BLOCKS; ++i) {
        vcm += s_vcm[i];
    }
    vcm /= npart;

    for (uint i = GTID; i < npart; i += GTDIM) {
#ifdef USE_VERLET_DSFUN
        vector_type v(g_v[i], g_v[i + nplace]);
#else
        vector_type v = g_v[i];
#endif
        v -= vcm;
        g_v[i] = static_cast<vector<float, dimension> >(v);
#ifdef USE_VERLET_DSFUN
        g_v[i + nplace] = dsfloat2lo(v);
#endif
        vv += v * v;
    }
    // reduced value for this thread
    s_vv[TID] = vv;
    __syncthreads();

    // compute reduced value for all threads in block
    reduce<THREADS / 2, sum_>(vv, s_vv);

    if (TID < 1) {
        // store block reduced value in global memory
        g_vv[blockIdx.x] = vv;
    }
}

/**
 * rescale velocities to accurate temperature
 */
template <typename vector_type, typename T>
__global__ void scale_velocity(T* g_v, uint npart, uint nplace, dsfloat const* g_vv, dsfloat temp)
{
    enum { dimension = vector_type::static_size };
    __shared__ dsfloat s_vv[THREADS];
    dsfloat vv = 0;

    // compute squared velocity sum from block reduced values
    for (uint i = TID; i < BLOCKS; i += TDIM) {
        s_vv[i] = g_vv[i];
    }
    __syncthreads();
    for (uint i = 0; i < BLOCKS; ++i) {
        vv += s_vv[i];
    }

    int dim = vector_type::static_size;
    dsfloat coeff = sqrt(temp * static_cast<dsfloat>(dim) * (static_cast<dsfloat>(npart) / vv));

    for (uint i = GTID; i < npart; i += GTDIM) {
#ifdef USE_VERLET_DSFUN
        vector_type v(g_v[i], g_v[i + nplace]);
#else
        vector_type v = g_v[i];
#endif
        v *= coeff;
        g_v[i] = static_cast<vector<float, dimension> >(v);
#ifdef USE_VERLET_DSFUN
        g_v[i + nplace] = dsfloat2lo(v);
#endif
    }
}

}}} // namespace halmd::cu::boltzmann

namespace halmd { namespace gpu
{

#ifdef USE_VERLET_DSFUN
typedef dsfloat float_type;
#else
typedef float float_type;
#endif

/**
 * device symbol wrappers
 */
cuda::symbol<uint48>
    boltzmann<>::rand48::a(cu::boltzmann::rand48::a);
cuda::symbol<uint48>
    boltzmann<>::rand48::c(cu::boltzmann::rand48::c);
cuda::symbol<ushort3*>
    boltzmann<>::rand48::state(cu::boltzmann::rand48::g_state);

/**
 * device function wrappers
 */

cuda::function<void (float4*, uint, uint, float, float4*)>
    boltzmann<3>::gaussian(cu::boltzmann::gaussian<cu::vector<float_type, 3> >);
cuda::function<void (float4*, uint, uint, float4 const*, dsfloat*)>
    boltzmann<3>::shift_velocity(cu::boltzmann::shift_velocity<cu::vector<float_type, 3> >);
cuda::function<void (float4*, uint, uint, dsfloat const*, dsfloat)>
    boltzmann<3>::scale_velocity(cu::boltzmann::scale_velocity<cu::vector<float_type, 3> >);

cuda::function<void (float2*, uint, uint, float, float2*)>
    boltzmann<2>::gaussian(cu::boltzmann::gaussian<cu::vector<float_type, 2> >);
cuda::function<void (float2*, uint, uint, float2 const*, dsfloat*)>
    boltzmann<2>::shift_velocity(cu::boltzmann::shift_velocity<cu::vector<float_type, 2> >);
cuda::function<void (float2*, uint, uint, dsfloat const*, dsfloat)>
    boltzmann<2>::scale_velocity(cu::boltzmann::scale_velocity<cu::vector<float_type, 2> >);

}} // namespace halmd::gpu
