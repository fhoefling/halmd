#include "hip/hip_runtime.h"
/* Lennard-Jones fluid kernel
 *
 * Copyright (C) 2008  Peter Colberg
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include "ljfluid_simple_glue.hpp"
#include "ljfluid_base.cu"
#include "algorithm.h"
#include "cutil.h"
#include "dsfun.h"
#include "vector2d.h"
#include "vector3d.h"

/**
 * MD simulation step
 */
template <typename T, typename U>
__global__ void mdstep(U* g_r, U* g_v, U* g_f, float* g_en, float* g_virial)
{
    extern __shared__ T s_r[];

    // load particle associated with this thread
    T r = unpack(g_r[GTID]);
    T v = unpack(g_v[GTID]);

    // potential energy contribution
    float en = 0;
    // virial equation sum contribution
    float virial = 0;

#ifdef DIM_3D
    dfloat3 f(make_float3(0, 0, 0));
#else
    dfloat2 f(make_float2(0, 0));
#endif

    // iterate over all blocks
    for (unsigned int k = 0; k < gridDim.x; k++) {
	// load positions of particles within block
	s_r[TID] = unpack(g_r[k * blockDim.x + TID]);
	__syncthreads();

	// iterate over all particles within block
	for (unsigned int j = 0; j < blockDim.x; j++) {
	    // skip placeholder particles
	    if (k * blockDim.x + j >= npart)
		continue;
	    // skip identical particle
	    if (blockIdx.x == k && TID == j)
		continue;

	    // compute Lennard-Jones force with particle
	    compute_force(r, s_r[j], f, en, virial);
	}
	__syncthreads();
    }

    // second leapfrog step of integration of equations of motion
    leapfrog_full_step(v, f.f0);

    // store particle associated with this thread
    g_v[GTID] = pack(v);
    g_f[GTID] = pack(f.f0);
    g_en[GTID] = en;
    g_virial[GTID] = virial;
}


namespace mdsim { namespace gpu { namespace ljfluid
{

#ifdef DIM_3D
cuda::function<void (float4*, float4*, float4*, float*, float*)> mdstep(::mdstep<float3>);
#else
cuda::function<void (float2*, float2*, float2*, float*, float*)> mdstep(::mdstep<float2>);
#endif

}}} // namespace mdsim::gpu::ljfluid
