#include "hip/hip_runtime.h"
/*
 * Copyright © 2012  Peter Colberg
 *
 * This file is part of HALMD.
 *
 * HALMD is free software: you can redistribute it and/or modify
 * it under the terms of the GNU Lesser General Public License as
 * published by the Free Software Foundation, either version 3 of
 * the License, or (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General
 * Public License along with this program. If not, see
 * <http://www.gnu.org/licenses/>.
 */

#include <halmd/config.hpp> // HALMD_GPU_DOUBLE_PRECISION
#include <halmd/numeric/blas/fixed_vector.hpp>
#include <halmd/utility/gpu/thread.cuh>
#include <test/unit/numeric/blas/fixed_vector_cuda_vector_converter_kernel.hpp>

using namespace halmd;

template <typename U, typename V>
static __global__ void converter_one(float4* g_input, float4* g_output, U* g_u, V* g_v)
{
    U u;
    V v;

    // pass by non-const reference
    tie(u, v) <<= g_input[GTID];
    g_output[GTID] <<= tie(u, v);

    // pass by const reference
    tie(u, v) <<= static_cast<float4 const&>(g_input[GTID]);
    g_output[GTID] <<= tie(static_cast<U const&>(u), static_cast<V const&>(v));

    // pass by value
    tie(u, v) <<= float4(g_input[GTID]);
    g_output[GTID] <<= make_tuple(u, v);

    g_u[GTID] = u;
    g_v[GTID] = v;
}

template <typename U, typename V>
static __global__ void converter_two(float4* g_input, float4* g_output, U* g_u, V* g_v)
{
    U u;
    V v;

    // pass by non-const reference
    tie(u, v) <<= tie(g_input[GTID], g_input[GTID + GTDIM]);
    tie(g_output[GTID], g_output[GTID + GTDIM]) <<= tie(u, v);

    // pass by const reference
    tie(u, v) <<= tie(static_cast<float4 const&>(g_input[GTID]), static_cast<float4 const&>(g_input[GTID + GTDIM]));
    tie(g_output[GTID], g_output[GTID + GTDIM]) <<= tie(static_cast<U const&>(u), static_cast<V const&>(v));

    // pass by value
    tie(u, v) <<= make_tuple(g_input[GTID], g_input[GTID + GTDIM]);
    tie(g_output[GTID], g_output[GTID + GTDIM]) <<= make_tuple(u, v);

    g_u[GTID] = u;
    g_v[GTID] = v;
}

template <typename U, typename V>
float_kernel<U, V> float_kernel<U, V>::kernel = {
    &::converter_one
};

template <typename U, typename V>
double_kernel<U, V> double_kernel<U, V>::kernel = {
    &::converter_two
};

template class float_kernel<fixed_vector<float, 3>, int>;
template class float_kernel<fixed_vector<float, 2>, int>;
template class float_kernel<fixed_vector<float, 3>, unsigned int>;
template class float_kernel<fixed_vector<float, 2>, unsigned int>;
template class float_kernel<fixed_vector<float, 3>, float>;
template class float_kernel<fixed_vector<float, 2>, float>;

template class double_kernel<fixed_vector<dsfloat, 3>, int>;
template class double_kernel<fixed_vector<dsfloat, 2>, int>;
template class double_kernel<fixed_vector<dsfloat, 3>, unsigned int>;
template class double_kernel<fixed_vector<dsfloat, 2>, unsigned int>;
template class double_kernel<fixed_vector<dsfloat, 3>, float>;
template class double_kernel<fixed_vector<dsfloat, 2>, float>;
template class double_kernel<fixed_vector<dsfloat, 3>, dsfloat>;
template class double_kernel<fixed_vector<dsfloat, 2>, dsfloat>;

#ifdef HALMD_GPU_DOUBLE_PRECISION
template class double_kernel<fixed_vector<double, 3>, int>;
template class double_kernel<fixed_vector<double, 2>, int>;
template class double_kernel<fixed_vector<double, 3>, unsigned int>;
template class double_kernel<fixed_vector<double, 2>, unsigned int>;
template class double_kernel<fixed_vector<double, 3>, float>;
template class double_kernel<fixed_vector<double, 2>, float>;
template class double_kernel<fixed_vector<double, 3>, double>;
template class double_kernel<fixed_vector<double, 2>, double>;
#endif
