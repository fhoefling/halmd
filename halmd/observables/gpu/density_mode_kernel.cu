#include "hip/hip_runtime.h"
/*
 * Copyright © 2008-2019  Felix Höfling
 * Copyright © 2015       Nicolas Höft
 * Copyright © 2008-2011  Peter Colberg
 *
 * This file is part of HALMD.
 *
 * HALMD is free software: you can redistribute it and/or modify
 * it under the terms of the GNU Lesser General Public License as
 * published by the Free Software Foundation, either version 3 of
 * the License, or (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General
 * Public License along with this program. If not, see
 * <http://www.gnu.org/licenses/>.
 */

#include <boost/utility/enable_if.hpp>

#include <halmd/numeric/blas/blas.hpp>
#include <halmd/observables/gpu/density_mode_kernel.hpp>
#include <halmd/utility/gpu/thread.cuh>

#define MAX_BLOCK_SIZE 1024

namespace halmd {
namespace observables {
namespace gpu {
namespace density_mode_kernel {

// pass wavevectors via texture
template<int dimension>
struct wavevector
{
    // instantiate a separate texture for each aligned vector type
    typedef texture<typename density_mode_wrapper<dimension>::coalesced_vector_type> type;
    static type tex_;
};
// instantiate static members
template<int dimension> wavevector<dimension>::type wavevector<dimension>::tex_;

// recursive reduction function,
// terminate for threads=0
template <unsigned threads, typename T>
__device__ typename boost::disable_if_c<threads>::type
sum_reduce(T*, T*) {}

// reduce two array simultaneously by summation,
// size of a,b must be at least 2 * threads
template <unsigned threads, typename T>
__device__ typename boost::enable_if_c<threads>::type
sum_reduce(T* a, T* b)
{
    if (TID < threads) {
        a[TID] += a[TID + threads];
        b[TID] += b[TID + threads];
    }

    if (threads >= warpSize) {
        __syncthreads();
    }
    else {
        // on hardware of compute capability ≥ 7.0 (Volta),
        // warps are no longer guaranteed to be executed in lock-step
#if CUDART_VERSION >= 9000
        // select warp lanes with TID < threads,
        // fix compilation of operator<< for large values of 'threads'
        unsigned mask = (1U << (threads & (warpSize - 1))) - 1;
        __syncwarp(mask);
#else
        __syncthreads();    // only needed if the _hardware_ is Volta or later
#endif
    }

    // recursion ends by calling sum_reduce<0>
    sum_reduce<threads / 2>(a, b);
}


/* FIXME
typedef void (*sum_reduce_type)(float*, float*);
__device__ sum_reduce_type sum_reduce_select[] = {
    &sum_reduce<0>, &sum_reduce<1>, &sum_reduce<2>, &sum_reduce<4>,
    &sum_reduce<8>, &sum_reduce<16>, &sum_reduce<32>, &sum_reduce<64>,
    &sum_reduce<128>, &sum_reduce<256>
};
*/

// FIXME provide complex data type for CUDA

/**
 *  compute exp(i q·r) for each particle/wavevector pair
 *  and sum results wavevector-wise within a block
 *
 *  @returns block sums of sin(q·r), cos(q·r) for each wavevector
 */
template <typename vector_type, typename coalesced_vector_type>
__global__ void compute(
    coalesced_vector_type const* g_r
  , unsigned int const* g_idx, int npart
  , float* g_sin_block, float* g_cos_block, int nq
)
{
    enum { dimension = vector_type::static_size };

    __shared__ float sin_[MAX_BLOCK_SIZE];
    __shared__ float cos_[MAX_BLOCK_SIZE];

    // outer loop over wavevectors
    for (int i=0; i < nq; i++) {
        vector_type q = tex1Dfetch(wavevector<dimension>::tex_, i);
        sin_[TID] = 0;
        cos_[TID] = 0;
        for (int j = GTID; j < npart; j += GTDIM) {
            // retrieve particle position via index array
            unsigned int idx = g_idx[j];
            vector_type r = g_r[idx];

            float q_r = inner_prod(q, r);
            sin_[TID] += sin(q_r);
            cos_[TID] += cos(q_r);
        }
        __syncthreads();

        // accumulate results within block
        if (TDIM == 1024) sum_reduce<512>(sin_, cos_);
        else if (TDIM == 512) sum_reduce<256>(sin_, cos_);
        else if (TDIM == 256) sum_reduce<128>(sin_, cos_);
        else if (TDIM == 128) sum_reduce<64>(sin_, cos_);
        else if (TDIM == 64) sum_reduce<32>(sin_, cos_);
        else if (TDIM == 32) sum_reduce<16>(sin_, cos_);
        else if (TDIM == 16) sum_reduce<8>(sin_, cos_);
        else if (TDIM == 8) sum_reduce<4>(sin_, cos_);

        if (TID == 0) {
            g_sin_block[i * BDIM + BID] = sin_[0];
            g_cos_block[i * BDIM + BID] = cos_[0];
        }
        __syncthreads();    // FIXME needed here? would __syncwarp() be sufficient?
    }
}

/**
 *  reduce block sums for each wavevector separately
 *
 *  @param bdim  number of blocks (grid size) in the preceding call to compute()
 */
__global__ void finalise(
    float const* g_sin_block, float const* g_cos_block
  , float* g_sin, float* g_cos
  , int nq, int bdim)
{
    __shared__ float s_sum[MAX_BLOCK_SIZE];
    __shared__ float c_sum[MAX_BLOCK_SIZE];

    // outer loop over wavevectors, distributed over block grid
    for (int i = BID; i < nq; i += BDIM) {
        s_sum[TID] = 0;
        c_sum[TID] = 0;
        for (int j = TID; j < bdim; j += TDIM) {
            s_sum[TID] += g_sin_block[i * bdim + j];
            c_sum[TID] += g_cos_block[i * bdim + j];
        }
        __syncthreads();

        // accumulate results within block
        if (TDIM == 1024) sum_reduce<512>(s_sum, c_sum);
        else if (TDIM == 512) sum_reduce<256>(s_sum, c_sum);
        else if (TDIM == 256) sum_reduce<128>(s_sum, c_sum);
        else if (TDIM == 128) sum_reduce<64>(s_sum, c_sum);
        else if (TDIM == 64) sum_reduce<32>(s_sum, c_sum);
        else if (TDIM == 32) sum_reduce<16>(s_sum, c_sum);
        else if (TDIM == 16) sum_reduce<8>(s_sum, c_sum);
        else if (TDIM == 8) sum_reduce<4>(s_sum, c_sum);

        // store result in global memory
        if (TID == 0) {
            g_sin[i] = s_sum[0];
            g_cos[i] = c_sum[0];
        }
        __syncthreads();    // FIXME needed here?
    }
}

} // namespace density_mode_kernel

template <int dimension>
density_mode_wrapper<dimension> const density_mode_wrapper<dimension>::kernel = {
    density_mode_kernel::wavevector<dimension>::tex_
  , density_mode_kernel::compute<fixed_vector<float, dimension> >
  , density_mode_kernel::finalise
};

template class density_mode_wrapper<3>;
template class density_mode_wrapper<2>;

} // namespace gpu
} // namespace observables
} // namespace halmd
