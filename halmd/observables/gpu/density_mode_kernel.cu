#include "hip/hip_runtime.h"
/*
 * Copyright © 2008-2019  Felix Höfling
 * Copyright © 2021       Jaslo Ziska
 * Copyright © 2015       Nicolas Höft
 * Copyright © 2008-2011  Peter Colberg
 *
 * This file is part of HALMD.
 *
 * HALMD is free software: you can redistribute it and/or modify
 * it under the terms of the GNU Lesser General Public License as
 * published by the Free Software Foundation, either version 3 of
 * the License, or (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General
 * Public License along with this program. If not, see
 * <http://www.gnu.org/licenses/>.
 */

#include <halmd/algorithm/gpu/reduction.cuh>
#include <halmd/algorithm/gpu/transform.cuh>
#include <halmd/numeric/blas/blas.hpp>
#include <halmd/observables/gpu/density_mode_kernel.hpp>
#include <halmd/utility/gpu/thread.cuh>

using namespace halmd::algorithm::gpu;

namespace halmd {
namespace observables {
namespace gpu {
namespace density_mode_kernel {

// FIXME provide complex data type for CUDA

/**
 *  compute exp(i q·r) for each particle/wavevector pair
 *  and sum results wavevector-wise within a block
 *
 *  @returns block sums of sin(q·r), cos(q·r) for each wavevector
 */
template <int dimension>
__global__ void compute(
    hipTextureObject_t wavevector
  , float4 const* g_r
  , unsigned int const* g_idx, int npart
  , float2* g_rho_block, int nq
)
{
    typedef fixed_vector<float, 2> complex_type;    // replacement for std::complex
    typedef fixed_vector<float, dimension> vector_type;
    typedef typename density_mode_wrapper<dimension>::coalesced_vector_type coalesced_vector_type;

    complex_type rho_;

    // outer loop over wavevectors
    for (int i=0; i < nq; i++) {
        vector_type q = tex1Dfetch<coalesced_vector_type>(wavevector, i);
        rho_ = 0;
        for (int j = GTID; j < npart; j += GTDIM) {
            // retrieve particle position via index array
            unsigned int idx = g_idx[j];
            vector_type r = g_r[idx];

            float q_r = inner_prod(q, r);
            // FIXME for huge simulation boxes, it may be necessary to use the
            // double precision versions cos() and sin() here
            rho_[0] += cosf(q_r);
            rho_[1] += sinf(q_r);
        }

        // accumulate results within block
        reduce<sum_>(rho_);

        if (TID == 0) {
            g_rho_block[i * BDIM + BID] = rho_;
        }
    }
}

/**
 *  reduce block sums for each wavevector separately
 *
 *  @param bdim  number of blocks (grid size) in the preceding call to compute()
 */
__global__ void finalise(float2 const* g_rho_block, float2* g_rho, int nq, int bdim)
{
    typedef fixed_vector<float, 2> complex_type;    // replacement for std::complex

    // outer loop over wavevectors, distributed over block grid
    for (int i = BID; i < nq; i += BDIM) {
        complex_type rho_sum = 0;
        for (int j = TID; j < bdim; j += TDIM) {
            rho_sum += static_cast<complex_type>(g_rho_block[i * bdim + j]);
        }

        // accumulate results within block
        reduce<sum_>(rho_sum);

        // store result in global memory
        if (TID == 0) {
            g_rho[i] = rho_sum;
        }
    }
}

} // namespace density_mode_kernel

template <int dimension>
density_mode_wrapper<dimension> density_mode_wrapper<dimension>::kernel = {
    density_mode_kernel::compute<dimension>
  , density_mode_kernel::finalise
};

template class density_mode_wrapper<3>;
template class density_mode_wrapper<2>;

} // namespace gpu
} // namespace observables
} // namespace halmd
