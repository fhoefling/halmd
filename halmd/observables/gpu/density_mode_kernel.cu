#include "hip/hip_runtime.h"
/*
 * Copyright © 2008-2019  Felix Höfling
 * Copyright © 2021       Jaslo Ziska
 * Copyright © 2015       Nicolas Höft
 * Copyright © 2008-2011  Peter Colberg
 *
 * This file is part of HALMD.
 *
 * HALMD is free software: you can redistribute it and/or modify
 * it under the terms of the GNU Lesser General Public License as
 * published by the Free Software Foundation, either version 3 of
 * the License, or (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General
 * Public License along with this program. If not, see
 * <http://www.gnu.org/licenses/>.
 */

#include <halmd/algorithm/gpu/reduction.cuh>
#include <halmd/algorithm/gpu/transform.cuh>
#include <halmd/numeric/blas/blas.hpp>
#include <halmd/observables/gpu/density_mode_kernel.hpp>
#include <halmd/utility/gpu/thread.cuh>

using namespace halmd::algorithm::gpu;

namespace halmd {
namespace observables {
namespace gpu {
namespace density_mode_kernel {

// FIXME provide complex data type for CUDA

/**
 *  compute exp(i q·r) for each particle/wavevector pair
 *  and sum results wavevector-wise within a block
 *
 *  @returns block sums of sin(q·r), cos(q·r) for each wavevector
 */
template <int dimension>
__global__ void compute(
    hipTextureObject_t t_wavevector
  , float4 const* g_r
  , unsigned int const* g_idx, int npart
  , float* g_sin_block, float* g_cos_block, int nq
)
{
    typedef fixed_vector<float, dimension> vector_type;
    typedef typename density_mode_wrapper<dimension>::coalesced_vector_type coalesced_vector_type;

    float sin_;
    float cos_;

    // outer loop over wavevectors
    for (int i=0; i < nq; i++) {
        vector_type q = tex1Dfetch<coalesced_vector_type>(t_wavevector, i);
        sin_ = 0;
        cos_ = 0;
        for (int j = GTID; j < npart; j += GTDIM) {
            // retrieve particle position via index array
            unsigned int idx = g_idx[j];
            vector_type r = g_r[idx];

            float q_r = inner_prod(q, r);
            sin_ += sin(q_r);
            cos_ += cos(q_r);
        }

        // accumulate results within block
        reduce<sum_>(sin_);
        reduce<sum_>(cos_);

        if (TID == 0) {
            g_sin_block[i * BDIM + BID] = sin_;
            g_cos_block[i * BDIM + BID] = cos_;
        }
    }
}

/**
 *  reduce block sums for each wavevector separately
 *
 *  @param bdim  number of blocks (grid size) in the preceding call to compute()
 */
__global__ void finalise(
    float const* g_sin_block, float const* g_cos_block
  , float* g_sin, float* g_cos
  , int nq, int bdim)
{
    float s_sum;
    float c_sum;

    // outer loop over wavevectors, distributed over block grid
    for (int i = BID; i < nq; i += BDIM) {
        s_sum = 0;
        c_sum = 0;
        for (int j = TID; j < bdim; j += TDIM) {
            s_sum += g_sin_block[i * bdim + j];
            c_sum += g_cos_block[i * bdim + j];
        }

        // accumulate results within block
        reduce<sum_>(s_sum);
        reduce<sum_>(c_sum);

        // store result in global memory
        if (TID == 0) {
            g_sin[i] = s_sum;
            g_cos[i] = c_sum;
        }
    }
}

} // namespace density_mode_kernel

template <int dimension>
density_mode_wrapper<dimension> density_mode_wrapper<dimension>::kernel = {
    density_mode_kernel::compute<dimension>
  , density_mode_kernel::finalise
};

template class density_mode_wrapper<3>;
template class density_mode_wrapper<2>;

} // namespace gpu
} // namespace observables
} // namespace halmd
