#include "hip/hip_runtime.h"
/*
 * Copyright © 2008-2010  Peter Colberg
 *
 * This file is part of HALMD.
 *
 * HALMD is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <halmd/mdsim/gpu/box_kernel.cuh>
#include <halmd/mdsim/gpu/particle_kernel.cuh>
#include <halmd/numeric/blas/blas.hpp>
#include <halmd/observables/gpu/phase_space_kernel.hpp>
#include <halmd/utility/gpu/thread.cuh>

using namespace halmd::mdsim::gpu; //< namespace box_kernel

namespace halmd {
namespace observables {
namespace gpu {
namespace phase_space_kernel {

/** positions, types */
texture<float4> r_;
/** minimum image vectors */
texture<void> image_;
/** velocities, tags */
texture<float4> v_;

/**
 * sample phase space for all particle of a single species
 */
template <typename vector_type, typename T>
__global__ void sample(
    unsigned int const* g_reverse_tag
  , T* g_r
  , T* g_v
  , vector_type box_length
  , unsigned int npart
)
{
    using mdsim::gpu::particle_kernel::untagged;
    using mdsim::gpu::particle_kernel::tagged;

    enum { dimension = vector_type::static_size };
    typedef typename phase_space_wrapper<dimension>::coalesced_vector_type coalesced_vector_type;

    if (GTID < npart) {
        // permutation index
        uint const rtag = g_reverse_tag[GTID];
        // fetch particle from texture caches
        unsigned int tag, type;
        vector_type r, v;
        tie(r, type) = untagged<vector_type>(tex1Dfetch(r_, rtag));
        tie(v, tag) = untagged<vector_type>(tex1Dfetch(v_, rtag));
        // extend particle positions in periodic box
        vector_type image = tex1Dfetch(reinterpret_cast<texture<coalesced_vector_type>&>(image_), rtag);
        box_kernel::extend_periodic(r, image, box_length);
        // store particle in global memory
        g_r[GTID] = tagged(r, type);
        g_v[GTID] = tagged(v, type);
    }
}

/**
 * shift particle positions to range (-L/2, L/2)
 */
template <typename vector_type, typename coalesced_vector_type>
__global__ void reduce_periodic(
    unsigned int const* g_reverse_tag
  , float4* g_r
  , coalesced_vector_type* g_image
  , vector_type box_length
  , unsigned int npart
)
{
    using mdsim::gpu::particle_kernel::untagged;
    using mdsim::gpu::particle_kernel::tagged;

    enum { dimension = vector_type::static_size };

    if (GTID < npart) {
        unsigned int rtag = g_reverse_tag[GTID];
        vector_type r;
        unsigned int type;
        tie(r, type) = untagged<vector_type>(tex1Dfetch(r_, rtag));

        vector_type image = box_kernel::reduce_periodic(r, box_length);

        g_image[rtag] = image;
        g_r[rtag] = tagged(r, type);
    }
}

} // namespace phase_space_kernel

template <int dimension>
phase_space_wrapper<dimension> const phase_space_wrapper<dimension>::kernel = {
    phase_space_kernel::r_
  , phase_space_kernel::image_
  , phase_space_kernel::v_
  , phase_space_kernel::sample<fixed_vector<float, dimension> >
  , phase_space_kernel::reduce_periodic<fixed_vector<float, dimension> >
};

template class phase_space_wrapper<3>;
template class phase_space_wrapper<2>;

} // namespace observables
} // namespace gpu
} // namespace halmd
