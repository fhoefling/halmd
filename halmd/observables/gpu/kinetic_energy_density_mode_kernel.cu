#include "hip/hip_runtime.h"
/*
 * Copyright © 2008-2022  Felix Höfling
 * Copyright © 2021       Jaslo Ziska
 * Copyright © 2015       Nicolas Höft
 * Copyright © 2008-2011  Peter Colberg
 *
 * This file is part of HALMD.
 *
 * HALMD is free software: you can redistribute it and/or modify
 * it under the terms of the GNU Lesser General Public License as
 * published by the Free Software Foundation, either version 3 of
 * the License, or (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General
 * Public License along with this program. If not, see
 * <http://www.gnu.org/licenses/>.
 */

#include <halmd/algorithm/gpu/reduction.cuh>
#include <halmd/algorithm/gpu/transform.cuh>
#include <halmd/numeric/blas/blas.hpp>
#include <halmd/observables/gpu/kinetic_energy_density_mode_kernel.hpp>
#include <halmd/utility/gpu/thread.cuh>

using namespace halmd::algorithm::gpu;

namespace halmd {
namespace observables {
namespace gpu {
namespace kinetic_energy_density_mode_kernel {

// FIXME provide complex data type for CUDA

/**
 *  compute (mv²/2) exp(i q·r) for each particle/wavevector pair
 *  and sum results wavevector-wise within a block
 *
 *  @returns block sums of (mv²/2) sin(q·r), (mv²/2) cos(q·r) for each wavevector
 */
template <typename vector_type, typename coalesced_vector_type>
__global__ void compute(
    hipTextureObject_t t_wavevector
  , coalesced_vector_type const* g_r
  , coalesced_vector_type const* g_v
  , unsigned int const* g_idx, int npart
  , float* g_sin_block, float* g_cos_block, int nq
)
{
    enum { dimension = vector_type::static_size };

    float sin_;
    float cos_;

    // outer loop over wavevectors
    for (int i=0; i < nq; i++) {
        vector_type q = tex1Dfetch<coalesced_vector_type>(t_wavevector, i);
        sin_ = 0;
        cos_ = 0;
        for (int j = GTID; j < npart; j += GTDIM) {
            // retrieve particle position via index array
            unsigned int idx = g_idx[j];
            vector_type r = g_r[idx];
            vector_type v;
            float mass;
            tie(v, mass) <<= g_v[idx];

            float ekin = mass * inner_prod(v, v) / 2;
            float q_r = inner_prod(q, r);
            sin_ += ekin * sin(q_r);
            cos_ += ekin * cos(q_r);
        }

        // accumulate results within block
        reduce<complex_sum_>(sin_, cos_);

        if (TID == 0) {
            g_sin_block[i * BDIM + BID] = sin_;
            g_cos_block[i * BDIM + BID] = cos_;
        }
    }
}

/**
 *  reduce block sums for each wavevector separately
 *
 *  @param bdim  number of blocks (grid size) in the preceding call to compute()
 */
__global__ void finalise(
    float const* g_sin_block, float const* g_cos_block
  , float* g_sin, float* g_cos
  , int nq, int bdim)
{
    float s_sum;
    float c_sum;

    // outer loop over wavevectors, distributed over block grid
    for (int i = BID; i < nq; i += BDIM) {
        s_sum = 0;
        c_sum = 0;
        for (int j = TID; j < bdim; j += TDIM) {
            s_sum += g_sin_block[i * bdim + j];
            c_sum += g_cos_block[i * bdim + j];
        }

        // accumulate results within block
        reduce<complex_sum_>(s_sum, c_sum);

        // store result in global memory
        if (TID == 0) {
            g_sin[i] = s_sum;
            g_cos[i] = c_sum;
        }
    }
}

} // namespace density_mode_kernel

template <int dimension>
kinetic_energy_density_mode_wrapper<dimension> kinetic_energy_density_mode_wrapper<dimension>::kernel = {
    kinetic_energy_density_mode_kernel::compute<fixed_vector<float, dimension> >
  , kinetic_energy_density_mode_kernel::finalise
};

template class kinetic_energy_density_mode_wrapper<3>;
template class kinetic_energy_density_mode_wrapper<2>;

} // namespace gpu
} // namespace observables
} // namespace halmd
