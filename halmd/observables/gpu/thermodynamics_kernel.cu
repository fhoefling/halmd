#include "hip/hip_runtime.h"
/*
 * Copyright © 2016      Felix Höfling
 * Copyright © 2013-2015 Nicolas Höft
 * Copyright © 2012      Peter Colberg
 *
 * This file is part of HALMD.
 *
 * HALMD is free software: you can redistribute it and/or modify
 * it under the terms of the GNU Lesser General Public License as
 * published by the Free Software Foundation, either version 3 of
 * the License, or (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General
 * Public License along with this program. If not, see
 * <http://www.gnu.org/licenses/>.
 */

#include <halmd/mdsim/force_kernel.hpp>
#include <halmd/algorithm/gpu/reduce_kernel.cuh>
#include <halmd/mdsim/gpu/box_kernel.cuh>
#include <halmd/numeric/mp/dsfloat.hpp>
#include <halmd/observables/gpu/thermodynamics_kernel.hpp>
#include <halmd/mdsim/force_kernel.hpp>

namespace halmd {
namespace observables {
namespace gpu {

template <int dimension, typename float_type>
__device__ void kinetic_energy<dimension, float_type>::operator()(size_type i)
{
    fixed_vector<float, dimension> v;
    float mass;
    tie(v, mass) <<= tex1Dfetch<float4>(texture_, i);
    mv2_ += mass * inner_prod(v, v);
}

template <int dimension, typename float_type>
__device__ void total_force<dimension, float_type>::operator()(size_type i)
{
    fixed_vector<float, dimension> f = tex1Dfetch(force<dimension>::tex_, i);
    force_ += f;
}

template <int dimension, typename float_type>
__device__ void centre_of_mass<dimension, float_type>::operator()(typename iterator::value_type const& value)
{
    size_type i;
    fixed_vector<float, dimension> box_length;
    tie(i, box_length) = value;
    fixed_vector<float, dimension> r, v, img;
    unsigned int species;
    float mass;
    tie(r, species) <<= tex1Dfetch<float4>(position_texture_, i);
    tie(v, mass) <<= tex1Dfetch<float4>(velocity_texture_, i);
    img = tex1Dfetch<coalesced_vector_type>(image_texture_, i);
    mdsim::gpu::box_kernel::extend_periodic(r, img, box_length);
    mr_ += mass * r;
    m_ += mass;
}

template <int dimension, typename float_type>
__device__ void velocity_of_centre_of_mass<dimension, float_type>::operator()(size_type i)
{
    fixed_vector<float, dimension> v;
    float mass;
    tie(v, mass) <<= tex1Dfetch<float4>(texture_, i);
    mv_ += mass * v;
    m_ += mass;
}

template <typename float_type>
__device__ void potential_energy<float_type>::operator()(size_type i)
{
    en_pot_ += tex1Dfetch<float>(texture_, i);
}

template <int dimension, typename float_type>
__device__ void virial<dimension, float_type>::operator()(size_type i)
{
    typedef fixed_vector<float, dimension> stress_pot_diagonal;
    stress_pot_diagonal v;
    v = mdsim::read_stress_tensor_diagonal<stress_pot_diagonal>(texture_, i, stride_);
    // add trace of the potential part of the stress tensor
    for (int j = 0; j < dimension; ++j) {
        virial_ += v[j];
    }
}

template <int dimension, typename float_type>
__device__ void stress_tensor<dimension, float_type>::operator()(size_type i)
{
    fixed_vector<float, dimension> v;
    float mass;

    stress_tensor_ += mdsim::read_stress_tensor<stress_tensor_type>(stress_pot_texture_, i, stride_);
    tie(v, mass) <<= tex1Dfetch<float4>(velocity_texture_, i);
    // compute the kinetic part of the stress tensor
    stress_tensor_ += mass * mdsim::make_stress_tensor(v);
}

template class observables::gpu::kinetic_energy<3, dsfloat>;
template class observables::gpu::kinetic_energy<2, dsfloat>;
template class observables::gpu::total_force<3, dsfloat>;
template class observables::gpu::total_force<2, dsfloat>;
template class observables::gpu::centre_of_mass<3, dsfloat>;
template class observables::gpu::centre_of_mass<2, dsfloat>;
template class observables::gpu::velocity_of_centre_of_mass<3, dsfloat>;
template class observables::gpu::velocity_of_centre_of_mass<2, dsfloat>;
template class observables::gpu::potential_energy<dsfloat>;
template class observables::gpu::virial<3, dsfloat>;
template class observables::gpu::virial<2, dsfloat>;
template class observables::gpu::stress_tensor<3, dsfloat>;
template class observables::gpu::stress_tensor<2, dsfloat>;

} // namespace gpu
} // namespace observables

template class reduction_kernel<observables::gpu::kinetic_energy<3, dsfloat> >;
template class reduction_kernel<observables::gpu::kinetic_energy<2, dsfloat> >;
template class reduction_kernel<observables::gpu::total_force<3, dsfloat> >;
template class reduction_kernel<observables::gpu::total_force<2, dsfloat> >;
template class reduction_kernel<observables::gpu::centre_of_mass<3, dsfloat> >;
template class reduction_kernel<observables::gpu::centre_of_mass<2, dsfloat> >;
template class reduction_kernel<observables::gpu::velocity_of_centre_of_mass<3, dsfloat> >;
template class reduction_kernel<observables::gpu::velocity_of_centre_of_mass<2, dsfloat> >;
template class reduction_kernel<observables::gpu::potential_energy<dsfloat> >;
template class reduction_kernel<observables::gpu::virial<3, dsfloat> >;
template class reduction_kernel<observables::gpu::virial<2, dsfloat> >;
template class reduction_kernel<observables::gpu::stress_tensor<3, dsfloat> >;
template class reduction_kernel<observables::gpu::stress_tensor<2, dsfloat> >;

} // namespace halmd
