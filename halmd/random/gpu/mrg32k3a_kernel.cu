#include "hip/hip_runtime.h"
/*
 * Copyright © 2016 Manuel Dibak
 *
 * This file is part of HALMD.
 *
 * HALMD is free software: you can redistribute it and/or modify
 * it under the terms of the GNU Lesser General Public License as
 * published by the Free Software Foundation, either version 3 of
 * the License, or (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General
 * Public License along with this program. If not, see
 * <http://www.gnu.org/licenses/>.
 */

#include <halmd/random/gpu/mrg32k3a_kernel.hpp>
#include <halmd/utility/gpu/thread.cuh>

namespace halmd {
namespace random {
namespace gpu {
namespace mrg32k3a_kernel {

__global__ void seed(hiprandStateMRG32k3a* state, uint seed)
{
    unsigned int id = GTID;
    /* Each thread gets same seed, a different sequence
       number, no offset */
    hiprand_init(seed, id, 0, &state[id]);
}

} // namespace mrg32k3a_kernel

/**
 * CUDA C++ wrappers
 */
mrg32k3a_wrapper mrg32k3a_wrapper::kernel = {
  mrg32k3a_kernel::seed
};

} // namespace random
} // namespace gpu
} // namespace halmd
