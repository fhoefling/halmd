#include "hip/hip_runtime.h"
/*
 * Copyright © 2016      Manuel Dibak
 *
 * This file is part of HALMD.
 *
 * HALMD is free software: you can redistribute it and/or modify
 * it under the terms of the GNU Lesser General Public License as
 * published by the Free Software Foundation, either version 3 of
 * the License, or (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General
 * Public License along with this program. If not, see
 * <http://www.gnu.org/licenses/>.
 */

#include <cmath>

#include <halmd/mdsim/gpu/box_kernel.cuh>
#include <halmd/mdsim/gpu/integrators/brownian_kernel.hpp>
#include <halmd/numeric/mp/dsfloat.hpp>
#include <halmd/random/gpu/normal_distribution.cuh>
#include <halmd/random/gpu/random_number_generator.cuh>
#include <halmd/utility/gpu/thread.cuh>

namespace halmd {
namespace mdsim {
namespace gpu {
namespace integrators {
namespace brownian_kernel {

/** array of diffusion constants */
static texture<float2> param_;

template <
    int dimension
  , typename float_type
  , typename ptr_type
  , typename gpu_vector_type
  , typename rng_type
>
__global__ void integrate(
    ptr_type g_position
  , gpu_vector_type* g_image
  , gpu_vector_type const* g_force
  , float timestep
  , float temp
  , rng_type rng
  , unsigned int nparticle
  , fixed_vector<float, dimension> box_length
)
{
    typedef fixed_vector<float_type, dimension> vector_type;
    typedef fixed_vector<float, dimension> float_vector_type;

    // kernel execution parameters
    unsigned int const thread = GTID;
    unsigned int const nthread = GTDIM;

    // read position, species, image from global memory
    vector_type r;
    unsigned int species;
    float mass;

    float_type rng_disp_cache = 0;
    bool rng_disp_cached = false;

    //read random number generator state from global device memory
    typename rng_type::state_type state = rng[thread];

    for (unsigned int i = thread; i < nparticle; i += nthread) {
        // read position (do this either way because we need the species)
        tie(r, species) <<= g_position[i];

        // diffusion constants (we will need at least one of those)
        fixed_vector<float, 2> diff_const = tex1Dfetch(param_, species);

        vector_type f = static_cast<float_vector_type>(g_force[i]);

        float_type const diff_const_disp = diff_const[0];
        float_type const sigma_disp = sqrtf(2 * diff_const_disp * timestep);

        // draw Gaussian random vector
        vector_type dr;
        tie(dr[0], dr[1]) =  random::gpu::normal(rng, state, 0, sigma_disp);
        if (dimension == 3) {
            if (rng_disp_cached) {
                dr[2] = rng_disp_cache;
            } else {
                tie(dr[2], rng_disp_cache) = random::gpu::normal(rng, state, 0, sigma_disp);
            }
            rng_disp_cached = !rng_disp_cached;
        }

        // Brownian integration: Euler-Maruyama scheme
        r += dr + (diff_const * timestep / temperature) * f;

        // enforce periodic boundary conditions
        float_vector_type image = box_kernel::reduce_periodic(r, box_length);

        // store position and image (do this here because the orientation doesn't change the position or species)
        g_position[i] <<= tie(r, species);
        if (!(image == float_vector_type(0))) {
            g_image[i] = image + static_cast<float_vector_type>(g_image[i]);
        }
    }

    // store random number generator state in global device memory
    rng[thread] = state;
}

} // namespace brownian_kernel

template <int dimension, typename float_type, typename rng_type>
cuda::texture<float2> brownian_wrapper<dimension, float_type, rng_type>::param = brownian_kernel::param_;

template <int dimension, typename float_type, typename rng_type>
brownian_wrapper<dimension, float_type, rng_type> const
brownian_wrapper<dimension, float_type, rng_type>::kernel = {
    brownian_kernel::integrate<dimension, float_type, ptr_type>
};

// explicit instantiation
#ifdef USE_GPU_SINGLE_PRECISION
template class brownian_wrapper<2, float, random::gpu::rand48_rng>;
template class brownian_wrapper<3, float, random::gpu::rand48_rng>;
#endif
#ifdef USE_GPU_DOUBLE_SINGLE_PRECISION
template class brownian_wrapper<2, dsfloat, random::gpu::rand48_rng>;
template class brownian_wrapper<3, dsfloat, random::gpu::rand48_rng>;
#endif

} // namespace integrators
} // namespace gpu
} // namespace mdsim
} // namespace halmd
