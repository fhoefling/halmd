#include "hip/hip_runtime.h"
/*
 * Copyright © 2011-2014 Felix Höfling
 * Copyright © 2011-2012 Michael Kopp
 *
 * This file is part of HALMD.
 *
 * HALMD is free software: you can redistribute it and/or modify
 * it under the terms of the GNU Lesser General Public License as
 * published by the Free Software Foundation, either version 3 of
 * the License, or (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General
 * Public License along with this program. If not, see
 * <http://www.gnu.org/licenses/>.
 */

#include <halmd/mdsim/gpu/box_kernel.cuh>
#include <halmd/mdsim/gpu/integrators/euler_kernel.hpp>
#include <halmd/numeric/mp/dsfloat.hpp>
#include <halmd/utility/gpu/thread.cuh>

namespace halmd {
namespace mdsim {
namespace gpu {
namespace integrators {
namespace euler_kernel {

/**
 * Euler integration: @f$ r(t + \Delta t) = r(t) + v(t) \Delta t @f$
 *
 * @param g_position    positions
 * @param g_image       number of times the particle exceeded the box margin
 * @param g_velocity    velocities
 * @param g_force       forces
 * @param timestep      integration timestep
 * @param box_length    edge lengths of cuboid box
 */
template <int dimension, typename float_type, typename ptr_type, typename const_ptr_type, typename gpu_vector_type>
__global__ void integrate(
    ptr_type g_position
  , gpu_vector_type* g_image
  , const_ptr_type g_velocity
  , float timestep
  , fixed_vector<float, dimension> box_length
)
{
    typedef fixed_vector<float_type, dimension> vector_type;
    typedef fixed_vector<float, dimension> float_vector_type;

    // kernel execution parameters
    unsigned int const thread = GTID;

    // read position, species, velocity, mass, image from global memory
    vector_type r, v;
    unsigned int species;
    float mass;
    tie(r, species) <<= g_position[thread];
    tie(v, mass) <<= g_velocity[thread];

    // Euler integration
    r += v * timestep;
    // enforce periodic boundary conditions
    float_vector_type image = box_kernel::reduce_periodic(r, box_length);

    // store position, species, image in global memory
    g_position[thread] <<= tie(r, species);
    if (!(image == float_vector_type(0))) {
        g_image[thread] = image + static_cast<float_vector_type>(g_image[thread]);
    }
}

} // namespace euler_kernel

template <int dimension, typename float_type>
euler_wrapper<dimension, float_type> const euler_wrapper<dimension, float_type>::kernel = {
    euler_kernel::integrate<dimension, float_type, ptr_type, const_ptr_type>
};

// explicit instantiation
#ifdef USE_GPU_SINGLE_PRECISION
template class euler_wrapper<3, float>;
template class euler_wrapper<2, float>;
#endif
#ifdef USE_GPU_DOUBLE_SINGLE_PRECISION
template class euler_wrapper<3, dsfloat>;
template class euler_wrapper<2, dsfloat>;
#endif
} // namespace integrators
} // namespace gpu
} // namespace mdsim
} // namespace halmd
