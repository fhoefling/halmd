#include "hip/hip_runtime.h"
/*
 * Copyright © 2008-2010  Peter Colberg
 *
 * This file is part of HALMD.
 *
 * HALMD is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <boost/mpl/if.hpp>

#include <halmd/mdsim/gpu/integrators/verlet_kernel.cuh>
#include <halmd/mdsim/gpu/integrators/verlet_kernel.hpp>
#include <halmd/mdsim/gpu/particle_kernel.cuh>
#include <halmd/numeric/blas/blas.hpp>
#include <halmd/numeric/mp/dsfloat.hpp>
#include <halmd/utility/gpu/thread.cuh>

using namespace boost::mpl;
using namespace halmd::mdsim::gpu::particle_kernel;

namespace halmd {
namespace mdsim {
namespace gpu {
namespace integrators {
namespace verlet_kernel {

/** integration time-step */
static __constant__ float timestep_;

/**
 * First leapfrog half-step of velocity-Verlet algorithm
 */
template <
    typename vector_type
  , typename vector_type_
  , typename gpu_vector_type
>
__global__ void _integrate(
    float4* g_r
  , gpu_vector_type* g_image
  , float4* g_v
  , gpu_vector_type const* g_f
  , float const* g_mass
  , unsigned int ntype
  , vector_type_ box_length
)
{
    extern __shared__ float s_mass[];
    if (TID < ntype) {
        s_mass[TID] = g_mass[TID];
    }
    __syncthreads();

    unsigned int const i = GTID;
    unsigned int const threads = GTDIM;
    unsigned int type, tag;
    vector_type r, v;
#ifdef USE_VERLET_DSFUN
    tie(r, type) = untagged<vector_type>(g_r[i], g_r[i + threads]);
    tie(v, tag) = untagged<vector_type>(g_v[i], g_v[i + threads]);
#else
    tie(r, type) = untagged<vector_type>(g_r[i]);
    tie(v, tag) = untagged<vector_type>(g_v[i]);
#endif
    vector_type_ image = g_image[i];
    vector_type_ f = g_f[i];
    float mass = s_mass[type];

    integrate(r, image, v, f, mass, timestep_, box_length);

#ifdef USE_VERLET_DSFUN
    tie(g_r[i], g_r[i + threads]) = tagged(r, type);
    tie(g_v[i], g_v[i + threads]) = tagged(v, tag);
#else
    g_r[i] = tagged(r, type);
    g_v[i] = tagged(v, tag);
#endif
    g_image[i] = image;
}

/**
 * Second leapfrog half-step of velocity-Verlet algorithm
 */
template <
    typename vector_type
  , typename vector_type_
  , typename gpu_vector_type
>
__global__ void _finalize(
    float4 const* g_r
  , float4* g_v
  , gpu_vector_type const* g_f
  , float const* g_mass
  , unsigned int ntype
)
{
    extern __shared__ float s_mass[];
    if (TID < ntype) {
        s_mass[TID] = g_mass[TID];
    }
    __syncthreads();

    unsigned int const i = GTID;
    unsigned int const threads = GTDIM;
    unsigned int tag, type;
    vector_type v;
    vector_type_ _;
    tie(_, type) = untagged<vector_type_>(g_r[i]);
#ifdef USE_VERLET_DSFUN
    tie(v, tag) = untagged<vector_type>(g_v[i], g_v[i + threads]);
#else
    tie(v, tag) = untagged<vector_type>(g_v[i]);
#endif
    vector_type_ f = g_f[i];
    float mass = s_mass[type];

    finalize(v, f, mass, timestep_);

#ifdef USE_VERLET_DSFUN
    tie(g_v[i], g_v[i + threads]) = tagged(v, tag);
#else
    g_v[i] = tagged(v, tag);
#endif
}

} // namespace verlet_kernel

template <int dimension>
verlet_wrapper<dimension> const verlet_wrapper<dimension>::wrapper = {
    verlet_kernel::timestep_
#ifdef USE_VERLET_DSFUN
  , verlet_kernel::_integrate<fixed_vector<dsfloat, dimension>, fixed_vector<float, dimension> >
  , verlet_kernel::_finalize<fixed_vector<dsfloat, dimension>, fixed_vector<float, dimension> >
#else
  , verlet_kernel::_integrate<fixed_vector<float, dimension>, fixed_vector<float, dimension> >
  , verlet_kernel::_finalize<fixed_vector<float, dimension>, fixed_vector<float, dimension> >
#endif
};

template class verlet_wrapper<3>;
template class verlet_wrapper<2>;

} // namespace mdsim
} // namespace gpu
} // namespace integrators
} // namespace halmd
