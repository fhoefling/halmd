#include "hip/hip_runtime.h"
/*
 * Copyright © 2008-2011  Peter Colberg and Felix Höfling
 *
 * This file is part of HALMD.
 *
 * HALMD is free software: you can redistribute it and/or modify
 * it under the terms of the GNU Lesser General Public License as
 * published by the Free Software Foundation, either version 3 of
 * the License, or (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General
 * Public License along with this program. If not, see
 * <http://www.gnu.org/licenses/>.
 */

#include <halmd/mdsim/gpu/positions/lattice_kernel.hpp>
#include <halmd/mdsim/positions/lattice_primitive.hpp>
#include <halmd/numeric/blas/blas.hpp>
#include <halmd/numeric/mp/dsfloat.hpp>
#include <halmd/utility/gpu/thread.cuh>

namespace halmd {
namespace mdsim {
namespace gpu {
namespace positions {
namespace lattice_kernel {

template <typename vector_type, typename lattice_type>
__global__ void lattice(
    float4* g_r
  , unsigned int npart
  , float a
  , unsigned int skip
  , typename lattice_type::result_type offset
  , typename lattice_type::shape_type ncell
)
{
    enum { dimension = vector_type::static_size };
    unsigned int const threads = GTDIM;

    lattice_type const lattice(ncell);

    for (unsigned int i = GTID; i < npart; i += threads) {

        // load particle type
        vector_type r;
        unsigned int type;
#ifdef USE_VERLET_DSFUN
        tie(r, type) <<= tie(g_r[i], g_r[i + threads]);
#else
        tie(r, type) <<= g_r[i];
#endif

        // introduce a vacancy after every (skip - 1) particles
        uint nvacancies = (skip > 1) ? (i / (skip - 1)) : 0;

        // compute primitive lattice vector
        fixed_vector<float, dimension> e = lattice(i + nvacancies);

        // scale with lattice constant and shift origin of lattice to offset
        r = e * a + offset; //< cast sum to dsfloat-based type

#ifdef USE_VERLET_DSFUN
        tie(g_r[i], g_r[i + threads]) <<= tie(r, type);
#else
        g_r[i] <<= tie(r, type);
#endif
    }
}

} // namespace lattice_kernel

template <typename lattice_type>
lattice_wrapper<lattice_type> const lattice_wrapper<lattice_type>::kernel = {
#ifdef USE_VERLET_DSFUN
    lattice_kernel::lattice<fixed_vector<dsfloat, dimension>, lattice_type>
#else
    lattice_kernel::lattice<fixed_vector<float, dimension>, lattice_type>
#endif
};

template class lattice_wrapper<close_packed_lattice<fixed_vector<float, 3>, fixed_vector<unsigned int, 3> > >;
template class lattice_wrapper<close_packed_lattice<fixed_vector<float, 2>, fixed_vector<unsigned int, 2> > >;

} // namespace mdsim
} // namespace gpu
} // namespace positions
} // namespace halmd
