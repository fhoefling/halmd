#include "hip/hip_runtime.h"
/*
 * Copyright © 2008-2011  Peter Colberg
 *
 * This file is part of HALMD.
 *
 * HALMD is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <halmd/mdsim/gpu/box_kernel.cuh>
#include <halmd/mdsim/gpu/particle_kernel.cuh>
#include <halmd/mdsim/gpu/positions/phase_space_kernel.hpp>
#include <halmd/numeric/blas/blas.hpp>
#include <halmd/utility/gpu/thread.cuh>
#include <halmd/utility/gpu/variant.cuh>

using namespace halmd::mdsim::gpu::particle_kernel;
using namespace halmd::utility::gpu;

/** cuboid box edge length */
static __constant__ variant<map<pair<int_<3>, float3>, pair<int_<2>, float2> > > box_length_;

namespace halmd {
namespace mdsim {
namespace gpu {
namespace positions {
namespace phase_space_kernel {

/**
 * shift particle positions to range (-L/2, L/2)
 */
template <typename vector_type>
__global__ void reduce_periodic(float4* g_r)
{
    enum { dimension = vector_type::static_size };

    vector_type r;
    unsigned int type;
    tie(r, type) = untagged<vector_type>(g_r[GTID]);

    vector_type box_length = get<dimension>(box_length_);

    box_kernel::reduce_periodic(r, box_length);

    g_r[GTID] = tagged(r, type);
}

} // namespace phase_space_kernel

template <int dimension>
phase_space_wrapper<dimension> const phase_space_wrapper<dimension>::kernel = {
    get<dimension>(box_length_)
  , phase_space_kernel::reduce_periodic<fixed_vector<float, dimension> >
};

template class phase_space_wrapper<3>;
template class phase_space_wrapper<2>;

} // namespace mdsim
} // namespace gpu
} // namespace positions
} // namespace halmd
