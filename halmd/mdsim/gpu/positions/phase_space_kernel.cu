#include "hip/hip_runtime.h"
/*
 * Copyright © 2008-2011  Peter Colberg
 *
 * This file is part of HALMD.
 *
 * HALMD is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <halmd/mdsim/gpu/box_kernel.cuh>
#include <halmd/mdsim/gpu/particle_kernel.cuh>
#include <halmd/mdsim/gpu/positions/phase_space_kernel.hpp>
#include <halmd/numeric/blas/blas.hpp>
#include <halmd/utility/gpu/thread.cuh>

using namespace halmd::mdsim::gpu::particle_kernel;

namespace halmd {
namespace mdsim {
namespace gpu {
namespace positions {
namespace phase_space_kernel {

/**
 * shift particle positions to range (-L/2, L/2)
 *
 * FIXME move to box_kernel.cuh
 */
template <typename vector_type, typename coalesced_vector_type>
__global__ void reduce_periodic(
    float4* g_r
  , coalesced_vector_type* g_image
  , vector_type box_length
)
{
    enum { dimension = vector_type::static_size };

    vector_type r;
    unsigned int type;
    tie(r, type) = untagged<vector_type>(g_r[GTID]);

    vector_type image = box_kernel::reduce_periodic(r, box_length);

    g_image[GTID] = image;
    g_r[GTID] = tagged(r, type);
}

} // namespace phase_space_kernel

template <int dimension>
phase_space_wrapper<dimension> const phase_space_wrapper<dimension>::kernel = {
    phase_space_kernel::reduce_periodic<fixed_vector<float, dimension> >
};

template class phase_space_wrapper<3>;
template class phase_space_wrapper<2>;

} // namespace mdsim
} // namespace gpu
} // namespace positions
} // namespace halmd
