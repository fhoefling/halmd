#include "hip/hip_runtime.h"
/*
 * Copyright © 2008-2011 Peter Colberg
 * Copyright © 2014      Nicolas Höft
 *
 * This file is part of HALMD.
 *
 * HALMD is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <halmd/mdsim/gpu/box_kernel.cuh>
#include <halmd/mdsim/gpu/neighbours/from_binning_kernel.hpp>
#include <halmd/mdsim/gpu/particle_kernel.cuh>
#include <halmd/numeric/blas/blas.hpp>
#include <halmd/utility/gpu/thread.cuh>

namespace halmd {
namespace mdsim {
namespace gpu {
namespace neighbours {
namespace from_binning_kernel {

/** (cutoff lengths + neighbour list skin)² */
texture<float> rr_cut_skin_;
/** positions, tags */
texture<float4> r_;

/**
 * compute neighbour cell
 */
inline __device__ unsigned int compute_neighbour_cell(
    fixed_vector<int, 3> const& offset
  , fixed_vector<int, 3> const& ncell
)
{
    // cell belonging to this execution block
    fixed_vector<int, 3> cell;
    cell[0] = BID % ncell[0];
    cell[1] = (BID / ncell[0]) % ncell[1];
    cell[2] = BID / ncell[0] / ncell[1];
    // neighbour cell of this cell
    cell = element_mod(cell + ncell + offset, ncell);

    return (cell[2] * ncell[1] + cell[1]) * ncell[0] + cell[0];
}

inline __device__ unsigned int compute_neighbour_cell(
    fixed_vector<int, 2> const& offset
  , fixed_vector<int, 2> const& ncell
)
{
    // cell belonging to this execution block
    fixed_vector<int, 2> cell;
    cell[0] = BID % ncell[0];
    cell[1] = BID / ncell[0];
    // neighbour cell of this cell
    cell = element_mod(cell + ncell + offset, ncell);

    return cell[1] * ncell[0] + cell[0];
}

/**
 * update neighbour list with particles of given cell
 */
template <bool same_cell, typename vector_type, typename cell_size_type, typename cell_difference_type>
__device__ void update_cell_neighbours(
    cell_difference_type const& offset
  , cell_size_type const& ncell
  , unsigned int const* g_cell
  , vector_type const& r
  , unsigned int type
  , unsigned int ntype1
  , unsigned int ntype2
  , unsigned int const& n
  , unsigned int& count
  , unsigned int* g_neighbour
  , unsigned int neighbour_size
  , unsigned int neighbour_stride
  , vector_type const& box_length
)
{
    extern __shared__ unsigned int s_n[];
    unsigned int* const s_type = &s_n[blockDim.x];
    vector_type* const s_r = reinterpret_cast<vector_type*>(&s_n[2 * blockDim.x]);

    // shared memory barrier
    __syncthreads();

    // compute cell index
    unsigned int const cell = compute_neighbour_cell(offset, static_cast<cell_difference_type>(ncell));
    // load particles in cell
    unsigned int const n_ = g_cell[cell * blockDim.x + threadIdx.x];
    s_n[threadIdx.x] = n_;
    tie(s_r[threadIdx.x], s_type[threadIdx.x]) <<= tex1Dfetch(r_, n_);
    __syncthreads();

    if (n == particle_kernel::placeholder) return;

    for (unsigned int i = 0; i < blockDim.x; ++i) {
        // particle number of cell placeholder
        unsigned int const m = s_n[i];
        // skip placeholder particles
        if (m == particle_kernel::placeholder) break;
        // skip same particle
        if (same_cell && i == threadIdx.x) continue;

        // particle distance vector
        vector_type dr = r - s_r[i];
        // enforce periodic boundary conditions
        box_kernel::reduce_periodic(dr, box_length);
        // squared particle distance
        float rr = inner_prod(dr, dr);

        // enforce cutoff length with neighbour list skin
        float rr_cut_skin = tex1Dfetch(rr_cut_skin_, type * ntype2 + s_type[i]);
        if (rr <= rr_cut_skin && count < neighbour_size) {
            // scattered write to neighbour list
            g_neighbour[count * neighbour_stride + n] = m;
            // increment neighbour list particle count
            count++;
        }
    }
}

/**
 * update neighbour lists
 */
template <unsigned int dimension>
__global__ void update_neighbours(
    int* g_ret
  , unsigned int* g_neighbour
  , unsigned int neighbour_size
  , unsigned int neighbour_stride
  , unsigned int const* g_cell
  , unsigned int ntype1
  , unsigned int ntype2
  , fixed_vector<unsigned int, dimension> ncell
  , fixed_vector<float, dimension> box_length
)
{
    // load particle from cell placeholder
    unsigned int const n = g_cell[GTID];
    unsigned int type;
    fixed_vector<float, dimension> r;
    tie(r, type) <<= tex1Dfetch(r_, n);
    // number of particles in neighbour list
    unsigned int count = 0;

    //
    // The summation of all forces acting on a particle is the most
    // critical part of the simulation concerning longtime accuracy.
    //
    // Naively adding all forces with a single-precision operation is fine
    // with the Lennard-Jones potential using the N-squared algorithm, as
    // the force exhibits both a repulsive and an attractive part, and the
    // particles are more or less in random order. Thus, summing over all
    // forces comprises negative and positive summands in random order.
    //
    // With the WCA potential (Weeks-Chandler-Andersen, purely repulsive
    // part of the shifted Lennard-Jones potential) using the N-squared
    // algorithm, the center of mass velocity effectively stays zero if
    // the initial list of particles arranged on a lattice is randomly
    // permuted before simulation.
    // Using the cell algorithm with the WCA potential however results
    // in a continuously drifting center of mass velocity, independent
    // of the chosen simulation timestep.
    //
    // The reason for this behaviour lies in the disadvantageous summing
    // order: With a purely repulsive potential, the summed forces of a
    // single neighbour cell will more or less have the same direction.
    // Thus, when adding the force sums of all neighbour cells, we add
    // huge force sums which will mostly cancel each other out in an
    // equilibrated system, giving a small and very inaccurate total
    // force due to being limited to single-precision floating-point
    // arithmetic.
    //
    // Besides implementing the summation in double precision arithmetic,
    // choosing the order of summation over cells such that one partial
    // neighbour cell force sum is always followed by the sum of the
    // opposite neighbour cell softens the velocity drift.
    //

    fixed_vector<int, dimension> j;
    for (j[0] = -1; j[0] <= 1; ++j[0]) {
        for (j[1] = -1; j[1] <= 1; ++j[1]) {
            if (dimension == 3) {
                for (j[2] = -1; j[2] <= 1; ++j[2]) {
                    if (j[0] == 0 && j[1] == 0 && j[2] == 0) {
                        goto self;
                    }
                    // visit 26 neighbour cells, grouped into 13 pairs of mutually opposite cells
                    update_cell_neighbours<false>(j, ncell, g_cell, r, type, ntype1, ntype2, n, count, g_neighbour, neighbour_size, neighbour_stride, box_length);
                    update_cell_neighbours<false>(-j, ncell, g_cell, r, type, ntype1, ntype2, n, count, g_neighbour, neighbour_size, neighbour_stride, box_length);
                }
            }
            else {
                if (j[0] == 0 && j[1] == 0) {
                    goto self;
                }
                // visit 8 neighbour cells, grouped into 4 pairs of mutually opposite cells
                update_cell_neighbours<false>(j, ncell, g_cell, r, type, ntype1, ntype2, n, count, g_neighbour, neighbour_size, neighbour_stride, box_length);
                update_cell_neighbours<false>(-j, ncell, g_cell, r, type, ntype1, ntype2, n, count, g_neighbour, neighbour_size, neighbour_stride, box_length);
            }
        }
    }

self:
    update_cell_neighbours<true>(j, ncell, g_cell, r, type, ntype1, ntype2, n, count, g_neighbour, neighbour_size, neighbour_stride, box_length);

    // return failure if any neighbour list is fully occupied
    if (count == neighbour_size) {
        *g_ret = EXIT_FAILURE;
    }
}

} // namespace from_binning_kernel

template <int dimension>
from_binning_wrapper<dimension> from_binning_wrapper<dimension>::kernel = {
    from_binning_kernel::rr_cut_skin_
  , from_binning_kernel::r_
  , from_binning_kernel::update_neighbours<dimension>
};

template class from_binning_wrapper<3>;
template class from_binning_wrapper<2>;

} // namespace neighbours
} // namespace gpu
} // namespace mdsim
} // namespace halmd
