#include "hip/hip_runtime.h"
/*
 * Copyright © 2011-2012  Michael Kopp
 *
 * This file is part of HALMD.
 *
 * HALMD is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <halmd/algorithm/gpu/reduction.cuh>
#include <halmd/mdsim/gpu/box_kernel.cuh> // reduce_periodic
#include <halmd/mdsim/gpu/mobilities/oseen_kernel.hpp>
#include <halmd/mdsim/gpu/particle_kernel.cuh> // tagged/untagged
#include <halmd/numeric/blas/fixed_vector/operators.hpp> //inner_product()
#include <halmd/numeric/mp/dsfloat.hpp> // sqrt
#include <halmd/utility/gpu/thread.cuh> // TID


using namespace halmd::algorithm::gpu;
using namespace halmd::mdsim::gpu::particle_kernel;

//
// Compute velocities from forces via Oseen/Rotne Prager Tensor calculus
//

namespace halmd {
namespace mdsim {
namespace gpu {
namespace mobilities {
namespace oseen_kernel {



/**
  * compute interactive-mobility
  *
  * @tparam order order of precision in (r/a) (1,2: oseen, >3: rotne-prager)
  * \note It is passed as template parameter, so that the compiler can decide
  * whether to implement oseen or rotne-prager part.
  *
  * \note This function is (normally) inlined automatically.
  */
template<
    int order
  , typename vector_type
  , typename vector_type_
>
__device__ void interaction_mobility(
    vector_type const& that_position
  , vector_type const& that_force
  , vector_type const& this_position
  , vector_type_& this_velocity
  , vector_type const& box_length
  , float const radius
)
{
    vector_type dr = this_position - that_position ;

    // apply minimum image convention
    box_kernel::reduce_periodic(dr, box_length);

    float dr2 = inner_prod(dr, dr);
    float dr_norm = sqrtf(dr2);
    float b = radius / dr_norm;

    // to the actual oseen stuff
    if( order <= 2 ) { //oseen
        this_velocity += (that_force + (inner_prod(dr,that_force) / dr2) * dr) * 0.75f * b;
    }
    else if (order <= 4) { // rotne prager
        if( dr_norm < 2*radius ) { // close branch
            this_velocity += ( 1 - (9.f / 32) * dr_norm / radius ) * that_force + ( (3.f / 32) * inner_prod(dr, that_force) / (radius * dr_norm) ) * dr;
        }
        else { // default branch
            float b2 = b * b;
            this_velocity += ((0.75f + 0.5f * b2) * b) * that_force + ((0.75f - 1.5f * b2) * b * inner_prod(dr, that_force) / dr2) * dr;
        }
    }
}


/**
  * update velocities from positions using oseen tensor calculus
  *
  * Every thread computes velocity of one single (associated) particle;
  * thread GTID is responsible for which one (g_v[GTID]).
  *
  * Positions and velocities are computed in single-precision, only for the
  * summing-up of velocities dsfun-precision is used.
  *
  * @param g_r positions in global device momory
  * @param g_f forces in global device momory
  * @param g_v velocities in global device momory -- will be updated in this function!
  * @param npart number of particles
  * @param radius hydrodynamic radius
  * @param self_mobility 1/(6*pi*eta*a) with eta being viscosity and a being radius
  * @tparam order order of precision in (r/a) (1,2: oseen, >3: rotne-prager)
  * @tparam vector_type float-vector type with appropriate dimension
  * @tparam vector_type_ dsfloat-vector type with appropriate dimension. If USE_OSEEN_DSFUN is set: dsfun. Else: float.
  * @tparam gpu_vector_type either float4 in 3D or float2 in 2D. Enables coalesced storage of forces.
  *
  */
template<
    int order
  , typename vector_type      // necessary for tagging/untagging stuff
  , typename vector_type_     // dsfun
  , typename gpu_vector_type  // forces: gpu::vector<gpu_vector_type> (float4 in 3D, float2 in 2d)
>
__global__ void _compute_velocities(
    const float4* g_r
  , const gpu_vector_type* g_f
  , float4* g_v
  , const unsigned int npart
  , const vector_type box_length
  , const float radius
  , const float self_mobility
)
{
    // get information which thread this is and thus which particles are to be processed
    unsigned int const i = GTID; // thread ID within grid
    unsigned int const threads_block = TDIM; // threads per block
    unsigned int const threads_grid = GTDIM; // threads per grid

    /* shared memory for this block
     *
     * In order to decrease the necessity for threads to read data from the
     * global memory, each block has some shared memory. It reads \e threads
     * particle positions and forces from global memory and stores them in the
     * shared memory. Then the threads in a block compute the velocities
     * resulting from this information. Only in the next timestep, information
     * from the global memory is being requested.
     *
     * \note CUDA only allows one pointer to shared memory. Yet there is the
     * special construct \code extern __shared__ type name[]; \endcode which
     * will create a (the) pointer to shared memory.  For this to work, a
     * default-shared-size must be passed to CUDA. This is done via a size_t
     * parameter in cuda::configure(..). It's the optional third parameter. So
     * make sure that configure(..) is called properly in the cpp file.
     */
    extern __shared__ char s_mem[];
    //! position of other particles in shared memory
    float4* const s_positions = reinterpret_cast<float4*>(s_mem);
    //! forces of other particles in shared memory
    gpu_vector_type* const s_forces = reinterpret_cast<gpu_vector_type*>(&s_positions[threads_block]);

    // position of particle associated with this particular thread (single precision)
    //
    // Although for particles with i >= npart the following does not make sense
    // (as there are no positions to be fetched), it does not harm though. The
    // particle module creates vectors big enough so that this operation will
    // not fail. So for each thread connected to a ghost particle there will be
    // one superfluous access to global memory. However if there was an
    // if-statement [if(i < npart)] before this, there would be one superflous
    // if statement for each single (real) particle. So as there are
    // (hopefully) much more real than ghost particles, it makes sense to
    // simply apply these operations to the ghost ones, too...
    //
    // Similar situations in this file will be denoted by a `[=*=]'-symbol.
    vector_type this_position = g_r[i];

    // velocity of particle associated with this particular thread
    vector_type_ this_velocity;
    unsigned int this_tag;
    // [=*=]
#ifdef USE_OSEEN_DSFUN
    tie(this_velocity, this_tag) = untagged<vector_type_>(g_v[i], g_v[i + threads_grid]);
#else
    tie(this_velocity, this_tag) = untagged<vector_type_>(g_v[i]);
#endif
    // reset velocity to zero //TODO this must be moved to a particle member function
    // Since we're in an overdamped regime, the velocity should consist solely
    // of external velocities (a `global' velocity). The velocity from the
    // previous timestep must not enter here.
    this_velocity = 0;

    // loop over every particle and consecutively add up velocity of this particle
    for(unsigned int tile_offset = 0; tile_offset < GTDIM; tile_offset+=TDIM) {
        // transfer positions and forces from global to shared memory
        s_positions[TID] = g_r[tile_offset + TID];
        s_forces[TID] = g_f[tile_offset + TID];
        __syncthreads(); //IMPORTANT: sync after reading. Otherwise a thread could request information not yet stored in shared memory.

        if( i < npart ) { //this could be removed [=*=]
            // loop over threads in this tile (= block)
            for(unsigned int k = 0; k < TDIM; ++k ) {
                if( tile_offset+k < npart ) { //IMPORTANT: this must not be removed!
                    // force on other particle
                    vector_type that_force = s_forces[k];

                    if( i == tile_offset+k ) { // self mobility
                        this_velocity += that_force;
                    }
                    else { // interaction
                        // position of other particle
                        vector_type that_position = s_positions[k];

                        // compute interaction of `this' and `that'
                        interaction_mobility<order>(that_position, that_force, this_position, this_velocity, box_length, radius);
                    }
                }
            }
        }
        __syncthreads(); //IMPORTANT: sync after computations
    }

    this_velocity *= self_mobility; // this has been factorized in previous computations

    // store final velocity for this particle [=*=]
#ifdef USE_OSEEN_DSFUN
    tie(g_v[i], g_v[i + threads_grid]) = tagged(this_velocity, this_tag);
#else
    g_v[i] = tagged(this_velocity, this_tag);
#endif

}


} // namespace oseen_kernel

template <int dimension>
oseen_wrapper<dimension> const oseen_wrapper<dimension>::wrapper = {
    /* gpu_vector_type does not have to be passed (in < >) as it's the type of
     * one argument and thus the compiler can identify is. On the other hand,
     * vector_type must be passed -- even though it's in the parameter list --,
     * because it's the first parameter and the _second_ (vector_type_) is
     * _not_ in the parameter list and thus has to be passed explicitly.
     *
     * This could be changed by simply using the order vector_type_,
     * vector_type, gpu_vector_type.
     */
#ifdef USE_OSEEN_DSFUN
    oseen_kernel::_compute_velocities<1, fixed_vector<float, dimension>, fixed_vector<dsfloat, dimension> > // _oseen
  , oseen_kernel::_compute_velocities<3, fixed_vector<float, dimension>, fixed_vector<dsfloat, dimension> > // _rotne
#else
    oseen_kernel::_compute_velocities<1, fixed_vector<float, dimension>, fixed_vector<float, dimension> > // _oseen
  , oseen_kernel::_compute_velocities<3, fixed_vector<float, dimension>, fixed_vector<float, dimension> > // _rotne
#endif
};

template class oseen_wrapper<3>;
template class oseen_wrapper<2>;

} // namespace mobilities
} // namespace gpu
} // namespace mdsim
} // namespace halmd
