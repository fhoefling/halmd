#include "hip/hip_runtime.h"
/* Copyright © 2019 Roya Ebrahimi Viand
 * Copyright © 2014-2015 Nicolas Höft
 *
 * This file is part of HALMD.
 *
 * HALMD is free software: you can redistribute it and/or modify
 * it under the terms of the GNU Lesser General Public License as
 * published by the Free Software Foundation, either version 3 of
 * the License, or (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General
 * Public License along with this program. If not, see
 * <http://www.gnu.org/licenses/>.
 */

#include <cub/iterator/counting_input_iterator.cuh>

#include <halmd/algorithm/gpu/copy_if_kernel.cuh>
#include <halmd/mdsim/geometries/cuboid.hpp>
#include <halmd/mdsim/geometries/sphere.hpp>
#include <halmd/mdsim/gpu/box_kernel.cuh>
#include <halmd/mdsim/gpu/particle_groups/region_species_kernel.hpp>
#include <halmd/utility/gpu/thread.cuh>

namespace halmd {
namespace mdsim {
namespace gpu {
namespace particle_groups{
namespace region_species_kernel {

/**
 *  Define selection criterion.
 */
template<typename geometry_type>
struct geometry_predicate
{
    enum { dimension = geometry_type::vector_type::static_size };
    typedef fixed_vector<float, dimension> vector_type;

    geometry_predicate(
        float4 const* position
      , geometry_type const& geometry
      , geometry_selection sel
      , unsigned int species
    )
      : position_(position)
      , geometry_(geometry)
      , selection_(sel)
      , species_(species)
    {};

    HALMD_GPU_ENABLED bool operator()(unsigned int i) const
    {
        vector_type r;
        unsigned int species;
        tie(r, species) <<= position_[i];
        bool in_species = (species == species_);
        bool in_geometry = geometry_(r);
        if (selection_ == excluded) {
            in_geometry = !in_geometry;
        }

        return (in_geometry && in_species);
    }

private:
    float4 const* position_; // position array
    geometry_type geometry_;
    geometry_selection selection_;
    unsigned int species_;
};

template <typename vector_type, typename geometry_type>
__global__ void compute_mask(
    float4 const* g_r
  , unsigned int nparticle
  , unsigned int* g_mask
  , geometry_type const geometry
  , geometry_selection selection
  , vector_type box_length
  , unsigned int species_
)
{
    enum { dimension = vector_type::static_size };
    unsigned int const i = GTID;
    if(i >= nparticle)
        return;

    vector_type r;
    unsigned int species;
    tie(r, species) <<= g_r[i];

    // enforce periodic boundary conditions
    box_kernel::reduce_periodic(r, box_length);
    bool in_species = (species == species_);
    bool in_geometry = geometry(r);
    if(selection == excluded)
        in_geometry = !in_geometry;
    // 1 means the particle in in the selector, 0 means outside
    g_mask[i] = (in_geometry && in_species) ? 1 : 0;
}

template <typename geometry_type>
unsigned int copy_selection(
    float4 const* g_r
  , unsigned int nparticle
  , unsigned int* g_output
  , geometry_type const geometry
  , geometry_selection selection
  , unsigned int species
)
{
    geometry_predicate<geometry_type> predicate(g_r, geometry, selection, species);

    // iterate over the particle indices, not the positions itself
    hipcub::CountingInputIterator<int> index(0);
    unsigned int output_size =
    halmd::algorithm::gpu::copy_if_kernel::copy_if(
        index
      , nparticle
      , predicate
      , g_output
    );

    return output_size;
}

} // namespace region_kernel

template<int dimension, typename geometry_type>
region_species_wrapper<dimension, geometry_type> const
region_species_wrapper<dimension, geometry_type>::kernel = {
    region_species_kernel::compute_mask
  , region_species_kernel::copy_selection<geometry_type>
};

template class region_species_wrapper<3, halmd::mdsim::geometries::cuboid<3, float> >;
template class region_species_wrapper<2, halmd::mdsim::geometries::cuboid<2, float> >;
template class region_species_wrapper<3, halmd::mdsim::geometries::sphere<3, float> >;
template class region_species_wrapper<2, halmd::mdsim::geometries::sphere<2, float> >;

} // namespace particle_groups
} // namespace gpu
} // namespace mdsim

} // namespace halmd
