#include "hip/hip_runtime.h"
/*
 * Copyright © 2008-2010  Peter Colberg and Felix Höfling
 *
 * This file is part of HALMD.
 *
 * HALMD is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <halmd/mdsim/gpu/forces/lj_kernel.hpp>
#include <halmd/mdsim/gpu/forces/pair_short_ranged_kernel.cuh>
#include <halmd/numeric/blas/blas.hpp>
#include <halmd/utility/gpu/variant.cuh>

namespace halmd
{
namespace mdsim { namespace gpu { namespace forces
{
namespace lj_kernel
{

/** global constants common to all truncated pair potentials */
__constant__ pair_short_ranged_kernel::_globals globals;
/** positions, types */
static texture<float4> r_;
/** array of Lennard-Jones potential parameters for all combinations of particle types */
static texture<float4> param_;

/** define Lennard-Jones potential */
struct lj_potential
{
    template <typename float_type, typename param_type>
    HALMD_GPU_ENABLED tuple<float_type, float_type> operator() (float_type rr, param_type const& param) const
    {
        float_type rri = param[SIGMA2] / rr;
        float_type ri6 = rri * rri * rri;
        float_type fval = 48 * param[EPSILON] * rri * ri6 * (ri6 - 0.5f) / param[SIGMA2];
        float_type en_pot = 4 * param[EPSILON] * ri6 * (ri6 - 1) - param[EN_CUT];

        return make_tuple(fval, en_pot);
    }

    HALMD_GPU_ENABLED texture<float4> const& param() const
    {
        return param_;
    }
};

template <typename vector_type, typename gpu_vector_type, typename stress_tensor_type>
__global__ void compute(
    gpu_vector_type* g_f
  , unsigned int* g_neighbour
  , float* g_en_pot
  , stress_tensor_type* g_stress_pot
)
{
    // call template function for truncated pair interactions
    pair_short_ranged_kernel::compute<vector_type>(
       lj_potential(), globals, r_
     , g_f, g_neighbour, g_en_pot, g_stress_pot
    );
}

} // namespace lj_kernel

template <int dimension>
lj_wrapper<dimension> const lj_wrapper<dimension>::kernel = {
    lj_kernel::compute<fixed_vector<float, dimension> >
  , get<dimension>(lj_kernel::globals.box_length)
  , lj_kernel::globals.neighbour_size
  , lj_kernel::globals.neighbour_stride
  , lj_kernel::r_
  , lj_kernel::param_
};

template class lj_wrapper<3>;
template class lj_wrapper<2>;

}}} // namespace mdsim::gpu::forces

} // namespace halmd

