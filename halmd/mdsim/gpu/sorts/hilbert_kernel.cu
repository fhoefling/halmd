#include "hip/hip_runtime.h"
/*
 * Copyright © 2008-2010  Peter Colberg
 *
 * This file is part of HALMD.
 *
 * HALMD is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <boost/mpl/if.hpp>
#include <float.h>

#include <halmd/algorithm/gpu/bits.cuh>
#include <halmd/mdsim/gpu/particle_kernel.cuh>
#include <halmd/mdsim/gpu/sorts/hilbert_kernel.hpp>
#include <halmd/mdsim/sorts/hilbert_kernel.hpp>
#include <halmd/numeric/blas/blas.hpp>
#include <halmd/utility/gpu/thread.cuh>
#include <halmd/utility/gpu/variant.cuh>

using namespace halmd::algorithm::gpu;
using namespace halmd::mdsim::gpu::particle_kernel;
using namespace halmd::utility::gpu;

namespace halmd {
namespace mdsim {
namespace gpu {
namespace sorts {
namespace hilbert_kernel {

/** Hilbert space-filling curve recursion depth */
__constant__ unsigned int depth_;
/** cubic box edgle length */
__constant__ variant<map<pair<int_<3>, float3>, pair<int_<2>, float2> > > box_length_;
/** positions, types */
texture<float4> r_;
/** minimum image vectors */
texture<variant<map<pair<int_<3>, float4>, pair<int_<2>, float2> > > > image_;
/** velocities, tags */
texture<float4> v_;

/**
 * generate Hilbert space-filling curve
 */
template <typename vector_type>
__global__ void map(float4 const* g_r, unsigned int* g_sfc)
{
    enum { dimension = vector_type::static_size };

    //
    // We need to avoid ambiguities during the assignment of a particle
    // to a subcell, i.e. the particle position should never lie on an
    // edge or corner of multiple subcells, or the algorithm will have
    // trouble converging to a definite Hilbert curve.
    //
    // Therefore, we use a simple cubic lattice of predefined dimensions
    // according to the number of cells at the deepest recursion level,
    // and round the particle position to the nearest center of a cell.
    //

    unsigned int type;
    vector_type r;
    tie(r, type) = untagged<vector_type>(g_r[GTID]);
    vector_type L = get<dimension>(box_length_);
    r = element_div(r, L);

    // compute Hilbert code for particle
    g_sfc[GTID] = mdsim::sorts::hilbert_kernel::map(r, depth_);
}

/**
 * generate ascending index sequence
 */
__global__ void gen_index(unsigned int* g_index)
{
    g_index[GTID] = GTID;
}

/**
 * order particles after given permutation
 */
template <typename vector_type, typename aligned_vector_type>
__global__ void order_particles(
    unsigned int const* g_index
  , float4* g_r
  , aligned_vector_type* g_image
  , float4* g_v
)
{
    enum { dimension = vector_type::static_size };

    unsigned int i = g_index[GTID];
    {
        vector_type r;
        unsigned int type;
#ifdef USE_VERLET_DSFUN
        tie(r, type) = untagged<vector_type>(tex1Dfetch(r_, i), tex1Dfetch(r_, i + GTDIM));
        tie(g_r[GTID], g_r[GTID + GTDIM]) = tagged(r, type);
#else
        tie(r, type) = untagged<vector_type>(tex1Dfetch(r_, i));
        g_r[GTID] = tagged(r, type);
#endif
    }
    {
        vector_type v;
        unsigned int tag;
#ifdef USE_VERLET_DSFUN
        tie(v, tag) = untagged<vector_type>(tex1Dfetch(v_, i), tex1Dfetch(v_, i + GTDIM));
        tie(g_v[GTID], g_v[GTID + GTDIM]) = tagged(v, tag);
#else
        tie(v, tag) = untagged<vector_type>(tex1Dfetch(v_, i));
        g_v[GTID] = tagged(v, tag);
#endif
    }
    g_image[GTID] = tex1Dfetch(get<dimension>(image_), i);
}

} // namespace hilbert_kernel

template <int dimension>
hilbert_wrapper<dimension> const hilbert_wrapper<dimension>::kernel = {
    hilbert_kernel::depth_
  , get<dimension>(hilbert_kernel::box_length_)
  , hilbert_kernel::r_
  , get<dimension>(hilbert_kernel::image_)
  , hilbert_kernel::v_
  , hilbert_kernel::map<fixed_vector<float, dimension> >
  , hilbert_kernel::gen_index
#ifdef USE_VERLET_DSFUN
  , hilbert_kernel::order_particles<fixed_vector<dsfloat, dimension> >
#else
  , hilbert_kernel::order_particles<fixed_vector<float, dimension> >
#endif
};

// explicit instantiation
template class hilbert_wrapper<3>;
template class hilbert_wrapper<2>;

} // namespace mdsim
} // namespace gpu
} // namespace sorts
} // namespace halmd
