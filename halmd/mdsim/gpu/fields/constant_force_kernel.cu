#include "hip/hip_runtime.h"
/*
 * Copyright © 2012  Michael Kopp
 *
 * This file is part of HALMD.
 *
 * HALMD is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <halmd/algorithm/gpu/apply_bind_kernel.cuh>
#include <halmd/algorithm/gpu/fill_kernel.cuh>
#include <halmd/numeric/blas/fixed_vector.hpp>

// Explicit instantiation of algorithms must happen in their namespace.
using namespace halmd::algorithm::gpu; // wrapper
using namespace halmd; // fixed_vector

// set in 3d
template class fill_wrapper<
    float4
  , float4
>;
// set in 2d
template class fill_wrapper<
    float2
  , float2
>;

// add in 3d
// Use fixed_vector, as it has operator+ defined.
template class apply_bind2nd_wrapper<
    sum_                                // functor
  , fixed_vector<float, 4>              // input_type
  , float4                              // coalesced_input_type
  , fixed_vector<float, 4>              // output_type
  , float4                              // coalesced_output_type
>;
// add in 2d
template class apply_bind2nd_wrapper<
    sum_                                // functor
  , fixed_vector<float, 2>              // input_type
  , float2                              // coalesced_input_type
  , fixed_vector<float, 2>              // output_type
  , float2                              // coalesced_output_type
>;
