#include "hip/hip_runtime.h"
/*
 * Copyright © 2012  Michael Kopp and Felix Höfling
 *
 * This file is part of HALMD.
 *
 * HALMD is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <halmd/algorithm/gpu/apply_bind_kernel.cuh>
#include <halmd/algorithm/gpu/fill_kernel.cuh>
#include <halmd/numeric/blas/fixed_vector.hpp>

using namespace halmd; // fixed_vector
using namespace halmd::algorithm::gpu; // kernel wrapper

// fill kernels
template class fill_preserve_tag_wrapper<
    fixed_vector<float, 3>              // value_type
>;
template class fill_preserve_tag_wrapper<
    fixed_vector<float, 2>              // value_type
>;

// fill kernels for high-precision part
template class fill_wrapper<
    fixed_vector<float, 3>              // value_type
  , float4                              // coalesced_value_type
>;
template class fill_wrapper<
    fixed_vector<float, 2>              // value_type
  , float4                              // coalesced_value_type
>;

// add constant value to array
template class apply_bind2nd_preserve_tag_wrapper<
    sum_                                // functor
  , fixed_vector<float, 3>              // input_type
>;
template class apply_bind2nd_preserve_tag_wrapper<
    sum_                                // functor
  , fixed_vector<float, 2>              // input_type
>;
