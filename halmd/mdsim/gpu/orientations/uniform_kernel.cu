#include "hip/hip_runtime.h"
/*
 * Copyright © 2016       Manuel Dibak
 * Copyright © 2008-2011  Peter Colberg and Felix Höfling
 *
 * This file is part of HALMD.
 *
 * HALMD is free software: you can redistribute it and/or modify
 * it under the terms of the GNU Lesser General Public License as
 * published by the Free Software Foundation, either version 3 of
 * the License, or (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General
 * Public License along with this program. If not, see
 * <http://www.gnu.org/licenses/>.
 */

#include <halmd/mdsim/gpu/orientations/uniform_kernel.hpp>
#include <halmd/mdsim/positions/lattice_primitive.hpp>
#include <halmd/numeric/blas/blas.hpp>
#include <halmd/numeric/mp/dsfloat.hpp>
#include <halmd/utility/gpu/thread.cuh>

namespace halmd {
namespace mdsim {
namespace gpu {
namespace orientations {
namespace uniform_kernel {

template <typename vector_type>
__global__ void uniform(
    float4* g_u
  , unsigned int npart
)
{
    enum { dimension = vector_type::static_size };
    unsigned int const threads = GTDIM;

    for (unsigned int i = GTID; i < npart; i += threads) {

        // load particle orientation
        vector_type u;
        unsigned int nothing;
#ifdef USE_VERLET_DSFUN
        tie(u, nothing) <<= tie(g_u[i], g_u[i + threads]);
#else
        tie(u, nothing) <<= g_u[i];
#endif
        u[0] = 1;
        u[1] = 0;
        u[2] = 0;

#ifdef USE_VERLET_DSFUN
        tie(g_u[i], g_u[i + threads]) <<= tie(u, nothing);
#else
        g_u[i] <<= tie(u, nothing);
#endif
    }
}

} // namespace uniform_kernel

uniform_wrapper const uniform_wrapper::kernel = {
#ifdef USE_VERLET_DSFUN
    uniform_kernel::uniform<fixed_vector<dsfloat, 3> >
#else
    uniform_kernel::uniform<fixed_vector<float, 3> >
#endif
};

//template class lattice_wrapper<close_packed_lattice<fixed_vector<float, 2>, fixed_vector<unsigned int, 2> > >;

} // namespace mdsim
} // namespace gpu
} // namespace orientations
} // namespace halmd
