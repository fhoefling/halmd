#include "hip/hip_runtime.h"
/*
 * Copyright © 2016       Manuel Dibak
 * Copyright © 2008-2011  Peter Colberg and Felix Höfling
 *
 * This file is part of HALMD.
 *
 * HALMD is free software: you can redistribute it and/or modify
 * it under the terms of the GNU Lesser General Public License as
 * published by the Free Software Foundation, either version 3 of
 * the License, or (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General
 * Public License along with this program. If not, see
 * <http://www.gnu.org/licenses/>.
 */

#include <halmd/mdsim/gpu/orientations/uniform_kernel.hpp>
#include <halmd/mdsim/positions/lattice_primitive.hpp>
#include <halmd/numeric/blas/blas.hpp>
#include <halmd/numeric/mp/dsfloat.hpp>
#include <halmd/random/gpu/random_number_generator.cuh>
#include <halmd/utility/gpu/thread.cuh>

#include <cmath>

namespace halmd {
namespace mdsim {
namespace gpu {
namespace orientations {
namespace uniform_kernel {

template <typename vector_type, typename rng_type>
__global__ void uniform(
    float4* g_u
  , unsigned int npart
  , rng_type rng
)
{
    enum { dimension = vector_type::static_size };
    unsigned int const threads = GTDIM;

    for (unsigned int i = GTID; i < npart; i += threads) {
        //read random number generator state from global device memory
        typename rng_type::state_type state = rng[GTID];

        // load particle orientation
        vector_type u;
        unsigned int nothing;
#ifdef USE_VERLET_DSFUN
        tie(u, nothing) <<= tie(g_u[i], g_u[i + threads]);
#else
        tie(u, nothing) <<= g_u[i];
#endif
        float theta = random::gpu::uniform(rng, state);
        float phi = random::gpu::uniform(rng, state);
        float pi = 4*atanf(1);

        theta =  acosf(2*theta - 1);
        phi = 2*pi*phi;

        /* select random point on unit sphere */ 
        u[0] = sin(theta) * cos(phi);
        u[1] = sin(theta) * sin(phi);
        u[2] = cos(theta);

#ifdef USE_VERLET_DSFUN
        tie(g_u[i], g_u[i + threads]) <<= tie(u, nothing);
#else
        g_u[i] <<= tie(u, nothing);
#endif
        rng[GTID] = state;
    }
}

} // namespace uniform_kernel

template <typename rng_type>
uniform_wrapper<rng_type> const uniform_wrapper<rng_type>::kernel = {
#ifdef USE_VERLET_DSFUN
    uniform_kernel::uniform<fixed_vector<dsfloat, 3>, rng_type>
#else
    uniform_kernel::uniform<fixed_vector<float, 3>, rng_type>
#endif
};

//template class lattice_wrapper<close_packed_lattice<fixed_vector<float, 2>, fixed_vector<unsigned int, 2> > >;

template class uniform_wrapper<random::gpu::rand48_rng>;
template class uniform_wrapper<random::gpu::mrg32k3a_rng>;

} // namespace mdsim
} // namespace gpu
} // namespace orientations
} // namespace halmd
