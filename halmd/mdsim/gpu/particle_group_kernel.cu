#include "hip/hip_runtime.h"
/*
 * Copyright © 2014 Nicolas Höft
 *
 * This file is part of HALMD.
 *
 * HALMD is free software: you can redistribute it and/or modify
 * it under the terms of the GNU Lesser General Public License as
 * published by the Free Software Foundation, either version 3 of
 * the License, or (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General
 * Public License along with this program. If not, see
 * <http://www.gnu.org/licenses/>.
 */

#include <halmd/mdsim/gpu/particle_group_kernel.hpp>
#include <halmd/utility/gpu/thread.cuh>
#include <halmd/utility/gpu/texture.cuh>
#include <halmd/utility/tuple.hpp>

/** positions, types */
static texture<float4> r_;
/** velocities, masses */
static texture<float4> v_;

namespace halmd {
namespace mdsim {
namespace gpu {
namespace particle_group_kernel {

/** positions, types */
static texture<float4> r_;
/** velocities, masses */
static texture<float4> v_;

/** minimum image vectors */
template<int dimension>
struct image
{
    // instantiate a separate texture for each aligned vector type
    typedef texture<typename particle_group_wrapper<float, dimension>::aligned_vector_type> type;
    static type tex_;
};
// instantiate static members
template<int dimension> image<dimension>::type image<dimension>::tex_;

/**
 * copy a subset of a particle instance (particle group) to another particle instance
 */
template <typename ptr_type, typename float_type, typename vector_type, typename aligned_vector_type>
__global__ void particle_group_to_particle(
    unsigned int const* g_index
  , ptr_type g_v
  , aligned_vector_type* g_image
  , ptr_type g_r
  , unsigned int npart
)
{
    enum { dimension = vector_type::static_size };

    if (GTID < npart) {
        int const i = g_index[GTID];

        // copy position and velocity as float4 values, and image vector
        g_r[GTID] = texFetch<float_type>::fetch(r_, i);
        g_v[GTID] = texFetch<float_type>::fetch(v_, i);

        // copy image vector with its type depending on the space dimension
        g_image[GTID] = tex1Dfetch(image<dimension>::tex_, i);
    }
}

} // namespace particle_group_kernel

template <typename float_type, int dimension>
particle_group_wrapper<float_type, dimension> const
particle_group_wrapper<float_type, dimension>::kernel = {
    particle_group_kernel::r_
  , particle_group_kernel::image<dimension>::tex_
  , particle_group_kernel::v_
  , particle_group_kernel::particle_group_to_particle<ptr_type, float_type, fixed_vector<float_type, dimension> >
};

template class particle_group_wrapper<float, 3>;
template class particle_group_wrapper<float, 2>;
template class particle_group_wrapper<dsfloat, 3>;
template class particle_group_wrapper<dsfloat, 2>;

} // namespace gpu
} // namespace mdsim
} // namespace halmd
