#include "hip/hip_runtime.h"
/*
 * Copyright © 2008-2011  Peter Colberg and Felix Höfling
 * Copyright © 2020       Jaslo Ziska
 *
 * This file is part of HALMD.
 *
 * HALMD is free software: you can redistribute it and/or modify
 * it under the terms of the GNU Lesser General Public License as
 * published by the Free Software Foundation, either version 3 of
 * the License, or (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General
 * Public License along with this program. If not, see
 * <http://www.gnu.org/licenses/>.
 */

#include <halmd/mdsim/gpu/forces/pair_full_kernel.cuh>
#include <halmd/mdsim/gpu/forces/pair_trunc_kernel.cuh>
#include <halmd/mdsim/gpu/potentials/pair/morse_kernel.hpp>
#include <halmd/mdsim/gpu/potentials/pair/truncations/truncations.cuh>
#include <halmd/numeric/blas/blas.hpp>

namespace halmd {
namespace mdsim {
namespace gpu {
namespace potentials {
namespace pair {
namespace morse_kernel {

__device__ void morse::fetch(
    unsigned int type1, unsigned int type2
  , unsigned int ntype1, unsigned int ntype2
)
{
    pair_ = tex1Dfetch<float4>(t_param_, type1 * ntype2 + type2);
}

} // namespace morse_kernel

HALMD_MDSIM_GPU_POTENTIALS_PAIR_TRUNCATIONS_INSTANTIATE_WRAPPERS(morse_kernel::morse);

} // namespace pair
} // namespace potentials

// explicit instantiation of force kernels
namespace forces {

using namespace halmd::mdsim::gpu::potentials::pair::morse_kernel;

template class pair_full_wrapper<3, morse>;
template class pair_full_wrapper<2, morse>;
HALMD_MDSIM_GPU_POTENTIALS_PAIR_TRUNCATIONS_INSTANTIATE_FORCE_KERNELS(morse);

} // namespace forces

} // namespace gpu
} // namespace mdsim
} // namespace halmd
