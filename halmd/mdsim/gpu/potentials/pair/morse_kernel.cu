#include "hip/hip_runtime.h"
/*
 * Copyright © 2008-2011  Peter Colberg and Felix Höfling
 *
 * This file is part of HALMD.
 *
 * HALMD is free software: you can redistribute it and/or modify
 * it under the terms of the GNU Lesser General Public License as
 * published by the Free Software Foundation, either version 3 of
 * the License, or (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General
 * Public License along with this program. If not, see
 * <http://www.gnu.org/licenses/>.
 */

#include <halmd/mdsim/gpu/forces/pair_full_kernel.cuh>
#include <halmd/mdsim/gpu/forces/pair_trunc_kernel.cuh>
#include <halmd/mdsim/gpu/potentials/pair/force_shifted_kernel.cuh>
#include <halmd/mdsim/gpu/potentials/pair/shifted_kernel.cuh>
#include <halmd/mdsim/gpu/potentials/pair/smooth_r4_kernel.cuh>
#include <halmd/mdsim/gpu/potentials/pair/morse_kernel.hpp>
#include <halmd/numeric/blas/blas.hpp>
#include <halmd/utility/tuple.hpp>

namespace halmd {
namespace mdsim {
namespace gpu {
namespace potentials {
namespace pair {
namespace morse_kernel {

/** array of potential parameters for all combinations of particle types */
static texture<float4> param_;

/**
 * Morse potential for the interaction of a pair of particles.
 */
class morse
{
public:
    /**
     * Construct Morse's pair interaction potential.
     *
     * Fetch potential parameters from texture cache for particle pair.
     *
     * @param type1 type of first interacting particle
     * @param type2 type of second interacting particle
     */
    HALMD_GPU_ENABLED morse(
        unsigned int type1, unsigned int type2
      , unsigned int ntype1, unsigned int ntype2
    )
      : pair_(tex1Dfetch(param_, type1 * ntype2 + type2))
    {}

    /**
     * Compute force and potential for interaction.
     *
     * @param rr squared distance between particles
     * @returns tuple of unit "force" @f$ -U'(r)/r @f$ and potential @f$ U(r) @f$
     */
    template <typename float_type>
    HALMD_GPU_ENABLED tuple<float_type, float_type> operator()(float_type rr) const
    {
        return morse_kernel::compute(rr, pair_[SIGMA], pair_[EPSILON], pair_[R_MIN_SIGMA]);
    }

private:
    /** potential parameters for particle pair */
    fixed_vector<float, 4> pair_;
};

} // namespace morse_kernel

cuda::texture<float4> morse_wrapper::param = morse_kernel::param_;
template class smooth_r4_wrapper<morse_kernel::morse>;
template class shifted_wrapper<morse_kernel::morse>;
template class force_shifted_wrapper<morse_kernel::morse>;

} // namespace pair
} // namespace potentials

// explicit instantiation of force kernels
namespace forces {

using namespace halmd::mdsim::gpu::potentials::pair::morse_kernel;
using namespace halmd::mdsim::gpu::potentials::pair::smooth_r4_kernel;
using namespace halmd::mdsim::gpu::potentials::pair::shifted_kernel;
using namespace halmd::mdsim::gpu::potentials::pair::force_shifted_kernel;

template class pair_full_wrapper<3, morse>;
template class pair_full_wrapper<2, morse>;
template class pair_trunc_wrapper<3, smooth_r4<morse> >;
template class pair_trunc_wrapper<2, smooth_r4<morse> >;
template class pair_trunc_wrapper<3, shifted<morse> >;
template class pair_trunc_wrapper<2, shifted<morse> >;
template class pair_trunc_wrapper<3, force_shifted<morse> >;
template class pair_trunc_wrapper<2, force_shifted<morse> >;

} // namespace forces

} // namespace gpu
} // namespace mdsim
} // namespace halmd
