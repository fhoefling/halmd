#include "hip/hip_runtime.h"
/*
 * Copyright © 2023 Felix Höfling
 *
 * This file is part of HALMD.
 *
 * HALMD is free software: you can redistribute it and/or modify
 * it under the terms of the GNU Lesser General Public License as
 * published by the Free Software Foundation, either version 3 of
 * the License, or (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General
 * Public License along with this program. If not, see
 * <http://www.gnu.org/licenses/>.
 */

#include <halmd/mdsim/gpu/forces/pair_full_kernel.cuh>
#include <halmd/mdsim/gpu/forces/pair_trunc_kernel.cuh>
#include <halmd/mdsim/gpu/potentials/pair/yukawa_kernel.hpp>
#include <halmd/mdsim/gpu/potentials/pair/truncations/truncations.cuh>
#include <halmd/numeric/blas/blas.hpp>

namespace halmd {
namespace mdsim {
namespace gpu {
namespace potentials {
namespace pair {
namespace yukawa_kernel {

__device__ void yukawa::fetch_param(
    unsigned int type1, unsigned int type2
  , unsigned int ntype1, unsigned int ntype2
)
{
    param_ = tex1Dfetch<float2>(t_param_, type1 * ntype2 + type2);
}

} // namespace yukawa_kernel

HALMD_MDSIM_GPU_POTENTIALS_PAIR_TRUNCATIONS_INSTANTIATE_WRAPPERS(yukawa_kernel::yukawa);

} // namespace pair
} // namespace potentials

// explicit instantiation of force kernels
namespace forces {

using namespace halmd::mdsim::gpu::potentials::pair::yukawa_kernel;

template class pair_full_wrapper<3, yukawa>;
template class pair_full_wrapper<2, yukawa>;
HALMD_MDSIM_GPU_POTENTIALS_PAIR_TRUNCATIONS_INSTANTIATE_FORCE_KERNELS(yukawa);

} // namespace forces

} // namespace gpu
} // namespace mdsim
} // namespace halmd
