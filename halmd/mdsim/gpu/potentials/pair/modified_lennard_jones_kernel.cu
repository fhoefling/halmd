#include "hip/hip_runtime.h"
/*
 * Copyright © 2008-2011  Peter Colberg and Felix Höfling
 *
 * This file is part of HALMD.
 *
 * HALMD is free software: you can redistribute it and/or modify
 * it under the terms of the GNU Lesser General Public License as
 * published by the Free Software Foundation, either version 3 of
 * the License, or (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General
 * Public License along with this program. If not, see
 * <http://www.gnu.org/licenses/>.
 */

#include <halmd/mdsim/gpu/forces/pair_full_kernel.cuh>
#include <halmd/mdsim/gpu/forces/pair_trunc_kernel.cuh>
#include <halmd/mdsim/gpu/potentials/pair/force_shifted_kernel.cuh>
#include <halmd/mdsim/gpu/potentials/pair/modified_lennard_jones_kernel.hpp>
#include <halmd/mdsim/gpu/potentials/pair/shifted_kernel.cuh>
#include <halmd/mdsim/gpu/potentials/pair/smooth_r4_kernel.cuh>
#include <halmd/numeric/blas/blas.hpp>
#include <halmd/numeric/pow.hpp>  // std::pow is not a device function
#include <halmd/utility/tuple.hpp>

namespace halmd {
namespace mdsim {
namespace gpu {
namespace potentials {
namespace pair {
namespace modified_lennard_jones_kernel {

/** array of Lennard-Jones potential parameters for all combinations of particle types */
static texture<float4> param_;

/**
 * Lennard-Jones interaction of a pair of particles.
 */
class modified_lennard_jones
{
public:
    /**
     * Construct Lennard-Jones pair interaction potential.
     *
     * Fetch potential parameters from texture cache for particle pair.
     *
     * @param type1 type of first interacting particle
     * @param type2 type of second interacting particle
     */
    HALMD_GPU_ENABLED modified_lennard_jones(
        unsigned int type1, unsigned int type2
      , unsigned int ntype1, unsigned int ntype2
    )
      : pair_(tex1Dfetch(param_, type1 * ntype2 + type2))
    {}

    /**
     * Compute force and potential for interaction.
     *
     * @param rr squared distance between particles
     * @returns tuple of unit "force" @f$ -U'(r)/r @f$ and potential @f$ U(r) @f$
     *
     * @f{eqnarray*}{
     *   - U'(r) / r &=& 4 r^{-2} \epsilon (\sigma/r)^{n} \left[ m (\sigma/r)^{m-n} - n \right] \\
     *   U(r) &=& 4 \epsilon (\sigma/r)^{n} \left[ (\sigma/r)^{m-n} - 1 \right]
     * @f}
     */
    template <typename float_type>
    HALMD_GPU_ENABLED tuple<float_type, float_type> operator()(float_type rr) const
    {
        return compute(rr, pair_[SIGMA2], pair_[EPSILON], static_cast<unsigned short>(pair_[INDEX_M_2])
                     , static_cast<unsigned short>(pair_[INDEX_N_2]));
    }

private:
    /** potential parameters for particle pair */
    fixed_vector<float, 4> pair_;
};

} // namespace modified_lennard_jones_kernel

cuda::texture<float4> modified_lennard_jones_wrapper::param = modified_lennard_jones_kernel::param_;
template class smooth_r4_wrapper<modified_lennard_jones_kernel::modified_lennard_jones>;
template class shifted_wrapper<modified_lennard_jones_kernel::modified_lennard_jones>;
template class force_shifted_wrapper<modified_lennard_jones_kernel::modified_lennard_jones>;

} // namespace pair
} // namespace potentials

// explicit instantiation of force kernels
namespace forces {

using namespace halmd::mdsim::gpu::potentials::pair::modified_lennard_jones_kernel;
using namespace halmd::mdsim::gpu::potentials::pair::smooth_r4_kernel;
using namespace halmd::mdsim::gpu::potentials::pair::shifted_kernel;
using namespace halmd::mdsim::gpu::potentials::pair::force_shifted_kernel;

template class pair_full_wrapper<3, modified_lennard_jones>;
template class pair_full_wrapper<2, modified_lennard_jones>;
template class pair_trunc_wrapper<3, smooth_r4<modified_lennard_jones> >;
template class pair_trunc_wrapper<2, smooth_r4<modified_lennard_jones> >;
template class pair_trunc_wrapper<3, shifted<modified_lennard_jones> >;
template class pair_trunc_wrapper<2, shifted<modified_lennard_jones> >;
template class pair_trunc_wrapper<3, force_shifted<modified_lennard_jones> >;
template class pair_trunc_wrapper<2, force_shifted<modified_lennard_jones> >;

} // namespace forces

} // namespace gpu
} // namespace mdsim
} // namespace halmd
