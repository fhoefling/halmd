#include "hip/hip_runtime.h"
/*
 * Copyright © 2008-2011  Peter Colberg and Felix Höfling
 *
 * This file is part of HALMD.
 *
 * HALMD is free software: you can redistribute it and/or modify
 * it under the terms of the GNU Lesser General Public License as
 * published by the Free Software Foundation, either version 3 of
 * the License, or (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General
 * Public License along with this program. If not, see
 * <http://www.gnu.org/licenses/>.
 */

#include <halmd/mdsim/gpu/forces/pair_full_kernel.cuh>
#include <halmd/mdsim/gpu/forces/pair_trunc_kernel.cuh>
#include <halmd/mdsim/gpu/potentials/pair/power_law_hard_core_kernel.cuh>
#include <halmd/mdsim/gpu/potentials/pair/power_law_kernel.hpp>
#include <halmd/mdsim/gpu/potentials/pair/adapters/truncations.cuh>
#include <halmd/numeric/blas/blas.hpp>
#include <halmd/numeric/pow.hpp>  // std::pow is not a device function
#include <halmd/utility/tuple.hpp>

namespace halmd {
namespace mdsim {
namespace gpu {
namespace potentials {
namespace pair {
namespace power_law_kernel {

/** array of potential parameters for all combinations of particle types */
static texture<float4> param_;

HALMD_GPU_ENABLED power_law::power_law(
    unsigned int type1, unsigned int type2
  , unsigned int ntype1, unsigned int ntype2
)
  : pair_(tex1Dfetch(param_, type1 * ntype2 + type2))
{}

template <typename float_type>
HALMD_GPU_ENABLED tuple<float_type, float_type> power_law::operator()(float_type rr) const
{
    return compute(rr, pair_[SIGMA2], pair_[EPSILON], static_cast<unsigned short>(pair_[INDEX]));
}

} // namespace power_law_kernel

cuda::texture<float4> power_law_wrapper::param = power_law_kernel::param_;
HALMD_MDSIM_GPU_POTENTIALS_PAIR_TRUNCATIONS_INSTANTIATE_WRAPPERS(power_law_kernel::power_law);

template class adapters::hard_core_wrapper<power_law_kernel::power_law>;
HALMD_MDSIM_GPU_POTENTIALS_PAIR_TRUNCATIONS_INSTANTIATE_WRAPPERS(
  adapters::hard_core_kernel::hard_core<power_law_kernel::power_law>
  );

} // namespace pair
} // namespace potentials

// explicit instantiation of force kernels
namespace forces {

using namespace halmd::mdsim::gpu::potentials::pair::power_law_kernel;
using namespace halmd::mdsim::gpu::potentials::pair::adapters::hard_core_kernel;

template class pair_full_wrapper<3, power_law>;
template class pair_full_wrapper<2, power_law>;
HALMD_MDSIM_GPU_POTENTIALS_PAIR_TRUNCATIONS_INSTANTIATE_FORCE_KERNELS(power_law);

template class pair_full_wrapper<3, hard_core<power_law> >;
template class pair_full_wrapper<2, hard_core<power_law> >;
HALMD_MDSIM_GPU_POTENTIALS_PAIR_TRUNCATIONS_INSTANTIATE_FORCE_KERNELS(hard_core<power_law>);

} // namespace forces

} // namespace gpu
} // namespace mdsim
} // namespace halmd
