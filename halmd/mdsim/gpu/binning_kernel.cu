#include "hip/hip_runtime.h"
/*
 * Copyright © 2008-2011  Peter Colberg
 *
 * This file is part of HALMD.
 *
 * HALMD is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <halmd/mdsim/gpu/binning_kernel.hpp>
#include <halmd/mdsim/gpu/particle_kernel.cuh>
#include <halmd/utility/gpu/thread.cuh>
#include <halmd/utility/gpu/variant.cuh>

using namespace halmd::mdsim::gpu::particle_kernel;
using namespace halmd::utility::gpu;

namespace halmd {
namespace mdsim {
namespace gpu {
namespace binning_kernel {

/** number of particles in simulation box */
__constant__ unsigned int nbox_;

/**
 * compute cell indices for given particle positions
 */
template <typename vector_type, typename cell_size_type>
inline __device__ unsigned int compute_cell_index(
    vector_type r
  , vector_type cell_length
  , cell_size_type ncell
)
{
    enum { dimension = vector_type::static_size };

    cell_size_type index = element_mod(
        static_cast<cell_size_type>(element_div(r, cell_length) + static_cast<vector_type>(ncell))
      , ncell
    );
    // FIXME check PTX to ensure CUDA unrolls this loop
    unsigned int offset = index[dimension - 1];
    for (int i = dimension - 2; i >= 0; i--) {
        offset *= ncell[i];
        offset += index[i];
    }
    return offset;
}

/**
 * compute cell indices for particle positions
 */
template <unsigned int dimension>
__global__ void compute_cell(
    float4 const* g_r
  , unsigned int* g_cell
  , fixed_vector<float, dimension> cell_length
  , fixed_vector<unsigned int, dimension> ncell
)
{
    fixed_vector<float, dimension> r;
    unsigned int type;
    tie(r, type) = untagged<fixed_vector<float, dimension> >(g_r[GTID]);
    g_cell[GTID] = compute_cell_index(r, cell_length, ncell);
}

/**
 * compute global cell offsets in particle list
 */
__global__ void find_cell_offset(unsigned int* g_cell, unsigned int* g_cell_offset)
{
    const unsigned int j = g_cell[GTID];
    const unsigned int k = (GTID > 0 && GTID < nbox_) ? g_cell[GTID - 1] : j;

    if (GTID == 0 || k < j) {
        // particle marks the start of a cell
        g_cell_offset[j] = GTID;
    }
}

/**
 * assign particles to cells
 */
__global__ void assign_cells(
  int* g_ret,
  unsigned int const* g_cell,
  unsigned int const* g_cell_offset,
  unsigned int const* g_itag,
  unsigned int* g_otag)
{
    __shared__ unsigned int s_offset[1];

    if (threadIdx.x == 0) {
        s_offset[0] = g_cell_offset[BID];
    }
    __syncthreads();
    // global offset of first particle in this block's cell
    const unsigned int offset = s_offset[0];
    // global offset of this thread's particle
    const unsigned int n = offset + threadIdx.x;
    // mark as virtual particle
    unsigned int tag = PLACEHOLDER;
    // mark as real particle if appropriate
    if (offset != PLACEHOLDER && n < nbox_ && g_cell[n] == BID) {
        tag = g_itag[n];
    }
    // return failure if any cell list is fully occupied
    if (tag != PLACEHOLDER && (threadIdx.x + 1) == blockDim.x) {
        *g_ret = EXIT_FAILURE;
    }
    // store particle in this block's cell
    g_otag[BID * blockDim.x + threadIdx.x] = tag;
}

/**
 * generate ascending index sequence
 */
__global__ void gen_index(unsigned int* g_index)
{
    g_index[GTID] = (GTID < nbox_) ? GTID : 0;
}

} // namespace binning_kernel

template <int dimension>
binning_wrapper<dimension> binning_wrapper<dimension>::kernel = {
    binning_kernel::nbox_
  , binning_kernel::assign_cells
  , binning_kernel::find_cell_offset
  , binning_kernel::gen_index
  , binning_kernel::compute_cell<dimension>
};

template class binning_wrapper<3>;
template class binning_wrapper<2>;

} // namespace gpu
} // namespace mdsim
} // namespace halmd
