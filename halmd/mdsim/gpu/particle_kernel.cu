#include "hip/hip_runtime.h"
/*
 * Copyright © 2010-2011 Felix Höfling
 * Copyright © 2015      Nicolas Höft
 * Copyright © 2010-2011 Peter Colberg
 *
 * This file is part of HALMD.
 *
 * HALMD is free software: you can redistribute it and/or modify
 * it under the terms of the GNU Lesser General Public License as
 * published by the Free Software Foundation, either version 3 of
 * the License, or (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General
 * Public License along with this program. If not, see
 * <http://www.gnu.org/licenses/>.
 */

#include <halmd/mdsim/gpu/particle_kernel.hpp>
#include <halmd/numeric/blas/blas.hpp>
#include <halmd/utility/gpu/thread.cuh>
#include <halmd/utility/tuple.hpp>

namespace halmd {
namespace mdsim {
namespace gpu {
namespace particle_kernel {

/** number of particles in simulation box */
static __constant__ unsigned int nbox_;
/** number of particle types */
static __constant__ unsigned int ntype_;
/** number of particles per type */
static texture<unsigned int> ntypes_;
/** positions, types */
static texture<float4> r_;
/** velocities, masses */
static texture<float4> v_;
/** IDs */
static texture<unsigned int> id_;

/** minimum image vectors */
template<int dimension>
struct image
{
    // instantiate a separate texture for each aligned vector type
    typedef texture<typename particle_wrapper<dimension>::aligned_vector_type> type;
    static type tex_;
};
// instantiate static members
template<int dimension> image<dimension>::type image<dimension>::tex_;

/**
 * rearrange particles by a given permutation
 */
template <typename vector_type, typename aligned_vector_type>
__global__ void rearrange(
    unsigned int const* g_index
  , float4* g_r
  , aligned_vector_type* g_image
  , float4* g_v
  , unsigned int* g_id
  , unsigned int npart
)
{
    enum { dimension = vector_type::static_size };
    if (GTID < npart) {
        int const i = g_index[GTID];

        // copy position and velocity as float4 values, and image vector
        g_r[GTID] = tex1Dfetch(r_, i);
        g_v[GTID] = tex1Dfetch(v_, i);

#ifdef USE_VERLET_DSFUN
        g_r[GTID + GTDIM] = tex1Dfetch(r_, i + GTDIM);
        g_v[GTID + GTDIM] = tex1Dfetch(v_, i + GTDIM);
#endif

        // select correct image texture depending on the space dimension
        g_image[GTID] = tex1Dfetch(image<dimension>::tex_, i);

        // copy particle IDs
        g_id[GTID] = tex1Dfetch(id_, i);
    }
}

} // namespace particle_kernel

template <int dimension>
particle_wrapper<dimension> const particle_wrapper<dimension>::kernel = {
    particle_kernel::nbox_
  , particle_kernel::ntype_
  , particle_kernel::ntypes_
  , particle_kernel::r_
  , particle_kernel::image<dimension>::tex_
  , particle_kernel::v_
  , particle_kernel::id_
#ifdef USE_VERLET_DSFUN
  , particle_kernel::rearrange<fixed_vector<dsfloat, dimension> >
#else
  , particle_kernel::rearrange<fixed_vector<float, dimension> >
#endif
};

template class particle_wrapper<3>;
template class particle_wrapper<2>;

} // namespace gpu
} // namespace mdsim
} // namespace halmd
