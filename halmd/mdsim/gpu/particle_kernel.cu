#include "hip/hip_runtime.h"
/*
 * Copyright © 2010-2011  Peter Colberg and Felix Höfling
 *
 * This file is part of HALMD.
 *
 * HALMD is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <halmd/algorithm/gpu/tuple.cuh>
#include <halmd/mdsim/gpu/particle_kernel.cuh>
#include <halmd/mdsim/gpu/particle_kernel.hpp>
#include <halmd/numeric/blas/blas.hpp>
#include <halmd/utility/gpu/thread.cuh>
#include <halmd/utility/gpu/variant.cuh>

using namespace halmd::algorithm::gpu;
using namespace halmd::utility::gpu;

namespace halmd {
namespace mdsim {
namespace gpu {
namespace particle_kernel {

/** number of particles in simulation box */
static __constant__ unsigned int nbox_;
/** number of particle types */
static __constant__ unsigned int ntype_;
/** number of particles per type */
static texture<unsigned int> ntypes_;
/** positions, types */
static texture<float4> r_;
/** minimum image vectors */
static texture<variant<map<pair<int_<3>, float4>, pair<int_<2>, float2> > > > image_;
/** velocities, tags */
static texture<float4> v_;

/**
 * set particle tags and types
 */
template <
    typename vector_type
  , typename coalesced_vector_type
>
__global__ void tag(coalesced_vector_type* g_r, coalesced_vector_type* g_v)
{
    vector_type r, v;
    unsigned int type, tag;
    tie(r, type) = untagged<vector_type>(g_r[GTID]);
    tie(v, tag) = untagged<vector_type>(g_v[GTID]);

    // set particle identifier unique within each particle type,
    // use a 0-based continuous numbering
    tag = GTID;

    // set particle type and adjust tag
    for (type = 0; type < ntype_; ++type) {
        unsigned int n = tex1Dfetch(ntypes_, type);
        if (tag < n) {
            break;
        }
        tag -= n;
    }

    g_r[GTID] = tagged(r, type);
    g_v[GTID] = tagged(v, tag);
}

/**
 * generate ascending index sequence
 */
__global__ void gen_index(unsigned int* g_index)
{
    g_index[GTID] = (GTID < nbox_) ? GTID : 0;
}

/**
 * rearrange particles by a given permutation
 */
template <typename vector_type, typename aligned_vector_type>
__global__ void rearrange(
    unsigned int const* g_index
  , float4* g_r
  , aligned_vector_type* g_image
  , float4* g_v
  , unsigned int* g_tag
)
{
    enum { dimension = vector_type::static_size };

    int const i = g_index[GTID];

    // copy position including type, and image vector
    g_r[GTID] = tex1Dfetch(r_, i);
#ifdef USE_VERLET_DSFUN
    g_r[GTID + GTDIM] = tex1Dfetch(r_, i + GTDIM);
#endif
    g_image[GTID] = tex1Dfetch(get<dimension>(image_), i);

    // copy velocity, but split off tag and store separately
    {
        vector_type v;
        unsigned int tag;
#ifdef USE_VERLET_DSFUN
        tie(v, tag) = untagged<vector_type>(tex1Dfetch(v_, i), tex1Dfetch(v_, i + GTDIM));
        tie(g_v[GTID], g_v[GTID + GTDIM]) = tagged(v, tag);
#else
        tie(v, tag) = untagged<vector_type>(tex1Dfetch(v_, i));
        g_v[GTID] = tagged(v, tag);
#endif
        g_tag[GTID] = tag;
    }
}

} // namespace particle_kernel

template <int dimension>
particle_wrapper<dimension> const particle_wrapper<dimension>::kernel = {
    particle_kernel::nbox_
  , particle_kernel::ntype_
  , particle_kernel::ntypes_
  , particle_kernel::r_
  , get<dimension>(particle_kernel::image_)
  , particle_kernel::v_
  , particle_kernel::tag<fixed_vector<float, dimension> >
  , particle_kernel::gen_index
#ifdef USE_VERLET_DSFUN
  , particle_kernel::rearrange<fixed_vector<dsfloat, dimension> >
#else
  , particle_kernel::rearrange<fixed_vector<float, dimension> >
#endif
};

template class particle_wrapper<3>;
template class particle_wrapper<2>;

} // namespace mdsim
} // namespace gpu
} // namespace halmd
